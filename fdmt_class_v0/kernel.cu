﻿
#include "hip/hip_runtime.h"
#include ""
#include "FdmtGpu.cuh"
#include <vector>
#include "npy.hpp"
#include "kernel.cuh"
#include <chrono>
#include "fileInput.h"
#include "DrawImg.h"
#include "Constants.h"
#include "FdmtCpu.h"

using namespace std;
enum TYPE_OF_PROCESSOR
{
	CPU
	, GPU
};

char strInpFolder[] = "..//FDMT_TESTS//2048";
char strPathOutImageNpyFile_gpu[] = "out_image_GPU.npy";
const bool BDIM_512_1024 = true;
TYPE_OF_PROCESSOR PROCESSOR = CPU;


void printDeviceInfo()
{
	int deviceId;
	int numberOfSMs;
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	hipDeviceProp_t deviceProps;
	hipGetDeviceProperties(&deviceProps, deviceId);
	std::string deviceName = deviceProps.name;
	std::cout << "Device Name: " << deviceName << std::endl;
	std::cout << "Number of SM: " << numberOfSMs << std::endl;
}


//---------------------------------------

int main(int argc, char** argv)
{
	printDeviceInfo();
	//--------------------------------------------------------------------------------------------------------------
	//------------------- prepare to work -------------------------------------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------------------

	// initiating input variables
	int iMaxDT = 0;
	int iImRows = 0, iImCols = 0;
	float val_fmax = 0., val_fmin = 0.;
	//int  nchan = iImRows;// 400;
	readDimensions(strInpFolder, &iImRows, &iImCols);
	// initiate pointer to input image
	fdmt_type_* h_parrImage = (fdmt_type_*)malloc(sizeof(fdmt_type_) * iImRows * iImCols);
	int ireturn = downloadInputData_gpu(strInpFolder, &iMaxDT, h_parrImage, &iImRows, &iImCols,
		&val_fmin, &val_fmax);

	fdmt_type_* u_parrImage = NULL;
	fdmt_type_* u_parrImOut = NULL;

	CFdmtB* pfdmt = nullptr;
	CFdmtGpu* pfdmt_gpu = nullptr;
	CFdmtCpu* pfdmt_Cpu = nullptr;
	if (PROCESSOR == GPU)
	{
		hipMalloc(&u_parrImage, sizeof(fdmt_type_) * iImRows * iImCols);
		hipMemcpy(u_parrImage, h_parrImage, sizeof(fdmt_type_) * iImRows * iImCols, hipMemcpyHostToDevice);
		hipMallocManaged(&u_parrImOut, iImCols * iMaxDT * sizeof(fdmt_type_));
		pfdmt = new CFdmtGpu(
			val_fmin
			, val_fmax
			, iImRows
			, iImCols
			, iMaxDT
		);
	}
	else
	{
		u_parrImage = (fdmt_type_*)malloc(sizeof(fdmt_type_) * iImRows * iImCols);
		memcpy(u_parrImage, h_parrImage, sizeof(fdmt_type_) * iImRows * iImCols);
		u_parrImOut = (fdmt_type_*)malloc(sizeof(fdmt_type_) * iMaxDT * iImCols);
		pfdmt = new CFdmtCpu(
			val_fmin
			, val_fmax
			, iImRows
			, iImCols
			, iMaxDT
		);
	}
	free(h_parrImage);
	
	iMaxDT = iMaxDT ;//iMaxDT * 3 +10;//iMaxDT//;	

	//--------------------------------------------------------------------------------------------------------------
	//-------------------- end of prepare ------------------------------------------------------------------------------------------
	//------------------- begin to calculate cuda var -------------------------------------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------------------
	//iImCols = 1 << 18;
	
	
	// 3. calculations		
	int num = 1000;
	auto start = std::chrono::high_resolution_clock::now();

	for (int i = 0; i < num; ++i)
	{
		pfdmt->process_image(u_parrImage       // on-device input image			
			, u_parrImOut	// OUTPUT image, dim = IMaxDT x IImgcols
			, false);
		
	}
	
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "PROCESSOR = "<< PROCESSOR<<"  Time taken by function fncFdmtU_cu : " << duration.count() / ((double)num) << " microseconds" << std::endl;

	delete pfdmt;
	// !3
	
	//4. write  output in .npy:IImgcols * IMaxDT * sizeof(int));
	hipDeviceSynchronize();
	fdmt_type_* parrImOut = (fdmt_type_*)malloc(iImCols * iMaxDT * sizeof(fdmt_type_));

	if (PROCESSOR == GPU)
	{		
		hipMemcpy(parrImOut, u_parrImOut, iImCols * iMaxDT * sizeof(fdmt_type_), hipMemcpyDeviceToHost);
	}
	else
	{
		memcpy(parrImOut, u_parrImOut, iImCols * iMaxDT * sizeof(fdmt_type_));
	}
	

	//std::vector<float> v1(parrImOut, parrImOut + iImCols * iMaxDT);
	std::vector<float> v1(iImCols * iMaxDT);
	for (int i = 0; i < iImCols * iMaxDT; ++i)
	{
		v1.at(i) = (float)parrImOut[i];
	}

	std::array<long unsigned, 2> leshape101{ iImCols ,iMaxDT};

	npy::SaveArrayAsNumpy(strPathOutImageNpyFile_gpu, false, leshape101.size(), leshape101.data(), v1);

	//--------------------------------------------------------------------------------------------------------------
	//-------------------- end of calculations ------------------------------------------------------------------------------------------
	//------------------- begin to draw output image for cuda -------------------------------------------------------------------------------------------

	float flops = 0;
	if (iImRows == 512)
	{
		flops = GFLPS_512;
	}
	else
	{
		if (iImRows == 1024)
		{
			if (BDIM_512_1024)
			{
				flops = GFLPS_512_1024;
			}
			else
			{
				flops = GFLPS_1024;
			}
		}
		else
		{
			flops = GFLPS_2048;
		}
	}

	cout << "GFLP/sec = " << ((double)flops) / ((double)duration.count() / ((double)num)) * 1.0e6 << endl;

	if (PROCESSOR == GPU)
	{
		hipFree(u_parrImage);
		hipFree(u_parrImOut);
	}
	else
	{
		free(u_parrImage);
		free(u_parrImOut);
	}
	free(parrImOut);		
	

	char filename_cpu[] = "image_gpu.png";
	createImg_(argc, argv, v1, iMaxDT, iImCols, filename_cpu);

	return 0;
}
