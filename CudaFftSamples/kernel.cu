#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


#include <complex>
#include <iostream>
#include <vector>
#include <hipfft/hipfft.h>

//#include "cufft_utils.h"
 // CUDA API error checking
//#ifndef CUDA_RT_CALL
//#define CUDA_RT_CALL( call )                                                                                           \
//    {                                                                                                                  \
//        auto status = static_cast<hipError_t>( call );                                                                \
//        if ( status != hipSuccess )                                                                                   \
//            fprintf( stderr,                                                                                           \
//                     "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "                                        \
//                     "with "                                                                                           \
//                     "%s (%d).\n",                                                                                     \
//                     #call,                                                                                            \
//                     __LINE__,                                                                                         \
//                     __FILE__,                                                                                         \
//                     hipGetErrorString( status ),                                                                     \
//                     status );                                                                                         \
//    }
//#endif  // CUDA_RT_CALL
//
//// cufft API error chekcing
//#ifndef CUFFT_CALL
//#define CUFFT_CALL( call )                                                                                             \
//    {                                                                                                                  \
//        auto status = static_cast<hipfftResult>( call );                                                                \
//        if ( status != HIPFFT_SUCCESS )                                                                                 \
//            fprintf( stderr,                                                                                           \
//                     "ERROR: CUFFT call \"%s\" in line %d of file %s failed "                                          \
//                     "with "                                                                                           \
//                     "code (%d).\n",                                                                                   \
//                     #call,                                                                                            \
//                     __LINE__,                                                                                         \
//                     __FILE__,                                                                                         \
//                     status );                                                                                         \
//    }
//#endif  // CUFFT_CALL

__global__
void scaling_kernel(hipfftComplex* data, int element_count, float scale) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride = blockDim.x * gridDim.x;
    for (auto i = tid; i < element_count; i += stride) {
        data[tid].x *= scale;
        data[tid].y *= scale;
    }
}


int main(int argc, char* argv[])
{
    /*char arrch[16] = { 1,2,3,4
    ,5,6,7,8
    ,9,10,11,12
    ,13,14,15,16 };
    hipfftComplex* d_arrch = nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_arrch), sizeof(char) * 16);
    hipMemcpyAsync(d_arrch, arrch, sizeof(char) * 16,  hipMemcpyHostToDevice);

    hipfftComplex* pcmparrRawSignalCur = nullptr;
    hipMallocManaged(reinterpret_cast<void**>(&pcmparrRawSignalCur), sizeof(hipfftComplex) * 16);


    hipMemcpy(pcmparrRawSignalCur, reinterpret_cast <hipfftComplex*> (d_arrch), 8 * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);

    for (int i = 0; i < 8; ++i)
    {
        std::printf("%f + %fj\n", pcmparrRawSignalCur[i].x, pcmparrRawSignalCur[i].y);
    }*/


    hipfftResult res;
    hipfftHandle plan;
    hipStream_t stream = NULL;

    int fft_size = 8;
    int batch_size = 1;// 2;
    int element_count = batch_size * fft_size;

    using scalar_type = float;
    using data_type = std::complex<scalar_type>;

    std::vector<data_type> data(element_count, 0);

    for (int i = 0; i < element_count; i++) {
        data[i] = data_type(i, -i);
    }

    std::printf("Input array:\n");
    for (auto& i : data) {
        std::printf("%f + %fj\n", i.real(), i.imag());
    }
    std::printf("=====\n");

    hipfftComplex* d_data = nullptr;
    res = hipfftCreate(&plan);
    if (res != HIPFFT_SUCCESS)
    {
        printf("ERROR: hipfftCreate failed\n");
    }
                                                               
    std::printf("fft_size = %i\n", fft_size);
    std::printf("batch_size = %i\n", batch_size);
    res = hipfftPlan1d(&plan, fft_size, HIPFFT_C2C, batch_size);

    if (res != HIPFFT_SUCCESS)
    {
        printf("ERROR: hipfftPlan1d failed\n");
    }
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    
    

    // Create device data arrays
    hipMalloc(reinterpret_cast<void**>(&d_data), sizeof(data_type) * data.size());
    hipMemcpyAsync(d_data, data.data(), sizeof(data_type) * data.size(),
        hipMemcpyHostToDevice, stream);

    /*
     * Note:
     *  Identical pointers to data and output arrays implies in-place transformation
     */
    res = hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    if (res != HIPFFT_SUCCESS)
    {
        printf("ERROR: hipfftPlan1d failed\n");
    }
    hipfftSetStream(plan, stream);
    std::printf("QU-QU !! \n");

    hipMemcpyAsync(data.data(), d_data, sizeof(data_type) * data.size(),
        hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    std::printf("Output array after Forward FFT :\n");
    for (auto& i : data) {
        std::printf("%f + %fj\n", i.real(), i.imag());
    }
    std::printf("=====\n");
    // Normalize the data
    scaling_kernel << <1, 128, 0, stream >> > (d_data, element_count, 1.f / fft_size);

    hipMemcpyAsync(data.data(), d_data, sizeof(data_type) * data.size(),
        hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    std::printf("Output array after Forward FFT, Normalization :\n");
    for (auto& i : data) {
        std::printf("%f + %fj\n", i.real(), i.imag());
    }
    std::printf("=====\n");

    // The original data should be recovered after Forward FFT, normalization and inverse FFT
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);
    

    hipMemcpyAsync(data.data(), d_data, sizeof(data_type) * data.size(),
        hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    std::printf("Output array after Forward FFT, Normalization, and Inverse FFT :\n");
    for (auto& i : data) {
        std::printf("%f + %fj\n", i.real(), i.imag());
    }
    std::printf("=====\n");

    /* free resources */
    hipFree(d_data);


    hipStreamDestroy(stream);

    hipDeviceReset();

    return EXIT_SUCCESS;
}


//
//int main(int argc, char* argv[]) {
//    hipfftHandle plan;
//    hipStream_t stream = NULL;
//
//    int fft_size = 8;
//    int batch_size = 2;
//    int element_count = batch_size * fft_size;
//
//    using scalar_type = float;
//    using data_type = std::complex<scalar_type>;
//
//    std::vector<data_type> data(element_count, 0);
//
//    for (int i = 0; i < element_count; i++) {
//        data[i] = data_type(i, -i);
//    }
//
//    std::printf("Input array:\n");
//    for (auto& i : data) {
//        std::printf("%f + %fj\n", i.real(), i.imag());
//    }
//    std::printf("=====\n");
//
//    hipfftComplex* d_data = nullptr;
//
//    CUFFT_CALL(hipfftCreate(&plan));
//    CUFFT_CALL(hipfftPlan1d(&plan, fft_size, HIPFFT_C2C, batch_size));
//
//    CUDA_RT_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
//    CUFFT_CALL(hipfftSetStream(plan, stream));
//
//    // Create device data arrays
//    CUDA_RT_CALL(hipMalloc(reinterpret_cast<void**>(&d_data), sizeof(data_type) * data.size()));
//    CUDA_RT_CALL(hipMemcpyAsync(d_data, data.data(), sizeof(data_type) * data.size(),
//        hipMemcpyHostToDevice, stream));
//
//    /*
//     * Note:
//     *  Identical pointers to data and output arrays implies in-place transformation
//     */
//    CUFFT_CALL(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
//
//    // Normalize the data
//    scaling_kernel << <1, 128, 0, stream >> > (d_data, element_count, 1.f / fft_size);
//
//    // The original data should be recovered after Forward FFT, normalization and inverse FFT
//    CUFFT_CALL(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD));
//
//    CUDA_RT_CALL(hipMemcpyAsync(data.data(), d_data, sizeof(data_type) * data.size(),
//        hipMemcpyDeviceToHost, stream));
//
//    CUDA_RT_CALL(hipStreamSynchronize(stream));
//
//    std::printf("Output array after Forward FFT, Normalization, and Inverse FFT :\n");
//    for (auto& i : data) {
//        std::printf("%f + %fj\n", i.real(), i.imag());
//    }
//    std::printf("=====\n");
//
//    /* free resources */
//    CUDA_RT_CALL(hipFree(d_data))
//
//        CUFFT_CALL(hipfftDestroy(plan));
//
//    CUDA_RT_CALL(hipStreamDestroy(stream));
//
//    CUDA_RT_CALL(hipDeviceReset());
//
//    return EXIT_SUCCESS;
//}

