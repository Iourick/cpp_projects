#include "hip/hip_runtime.h"
﻿#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#include "clusterization_test.cuh"
#include "Clusterization.cuh"
#include "Constants.h"


#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/sequence.h>

#include <thrust/gather.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include "select_delegates.cuh"
//----------------------------------------------------------------------------------------------------------------
// CUDA kernel to initialize arrays
__global__ void initialize_arrays(int* arrt, int* arrdt, int* arrwidth, int* arrsnr, int num_elements)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements)
    {
        arrt[idx] = num_elements - idx;
        arrdt[idx] = num_elements - idx;
        arrwidth[idx] = num_elements - idx;
        arrsnr[idx] = num_elements - idx;
    }
}
//----------------------------------------------------------------------------------
//---------------------------------------------------------------------------------------------
// CUDA kernel to initialize Cand vector
__global__ void initialize_cand(Cand* d_vec_cand, int* arrt, int* arrdt, int* arrwidth, int* arrsnr, int num_elements)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements)
    {
        d_vec_cand[idx].mt = arrt[idx];
        d_vec_cand[idx].mdt = arrdt[idx];
        d_vec_cand[idx].mwidth = arrwidth[idx];
    }
}
//---------------------------------------------------------------------------------------
int main() 
{
    // 1 Create arrays  and stuff them on GPU
    const int rows = 3;// 256;
    const int cols = 16;// (1 << 20) / rows;
    thrust::device_vector<fdmt_type_ >d_vctInp(rows * cols);
    fdmt_type_* d_arr = thrust::raw_pointer_cast(d_vctInp.data());
    
    thrust::device_vector<fdmt_type_ >d_vctNorm(rows * cols);

    fdmt_type_* d_norm = thrust::raw_pointer_cast(d_vctNorm.data());
    
    float* arr = (float*)malloc(rows * cols * sizeof(fdmt_type_));
    float* norm = (float*)malloc(rows * cols * sizeof(fdmt_type_));
    for (int i = 0; i < rows * cols; ++i)
    {
        arr[i] = 2;
        norm[i] = 4.;
    }
    arr[4] = 140.;
    arr[11] = 120.;
    arr[2 * cols +4] = 100.;
   
    
    for (int i = 3; i < 5; ++i)
    {
     //  norm[cols + i] = 0.001;
    }

    hipMemcpy(d_arr, arr, rows * cols * sizeof(fdmt_type_), hipMemcpyHostToDevice);
    hipMemcpy(d_norm, norm, rows * cols * sizeof(fdmt_type_), hipMemcpyHostToDevice);
    // !1

    // 2.max length of window
    const int WndWidth = 4;
    //!2

    // 3. treshold
    float valTresh_ = 4.;
    float* d_pTresh;     // Device pointer
   

    // Allocate memory on the GPU for the float variable
    hipError_t err = hipMalloc((void**)&d_pTresh, sizeof(float));
    hipMemcpy(d_pTresh, &valTresh_, sizeof(float), hipMemcpyHostToDevice);
   
    // !3

    // 4. metrics array on host and device
    thrust::host_vector<int> h_bin_metrics(3);
    h_bin_metrics[0] = 2;
    h_bin_metrics[1] = 1;
    h_bin_metrics[2] = 2;

    thrust::device_vector<int> d_bin_metrics( h_bin_metrics.size());
    d_bin_metrics = h_bin_metrics;
    // !4

    // 5. normalization 
    //thrust::device_vector<float> d_normalized_fdmt(rows * cols);
    //const int blocksize0 = 1024;
    //const int gridsize0 = (rows * cols + blocksize0 - 1) / blocksize0;    
    //normalize_fdmt_kernel << <gridsize0, blocksize0 >> > (d_arr, d_norm, rows * cols, thrust::raw_pointer_cast(d_normalized_fdmt.data()));
    //
    //auto max_iter = thrust::max_element(d_normalized_fdmt.begin(), d_normalized_fdmt.end());
    //float max_value = *max_iter;


    //// 6. digitizing normalized fdmt

    //// Create the d_digitized vector
    //thrust::device_vector<int> d_digitized(d_normalized_fdmt.size());

    //// Apply the transformation to digitize the values
    //thrust::transform(d_normalized_fdmt.begin(), d_normalized_fdmt.end(),
    //    d_digitized.begin(), DigitizeFunctor(max_value));

   // thrust::device_vector<int> d_fdmt_digitized(rows * cols);
    const int* d_pbin_metrics = thrust::raw_pointer_cast(d_bin_metrics.data());

    // start of work
    const int blocksize0 = 1024;
    clusterization::digitize_kernel << <1, blocksize0 , blocksize0  * sizeof(int) >> > (d_arr,  rows * cols);
   
   /* hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << "pipets" << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    bool are_equal = thrust::equal(d_fdmt_digitized.begin(), d_fdmt_digitized.end(), d_digitized.begin());

    int iarr[48] = { 0 }, iarr1[48] = { 0 };
    hipMemcpy(iarr1, thrust::raw_pointer_cast(d_fdmt_digitized.data()), 48 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 48; ++i)
    {
        iarr[i] = (int)(arr[i] /sqrtf(0.0001 + norm[i]) * 128.0 / 70.0);
    }*/
    /*clusterization(const int* d_digitized_fdmt
        , const int Rows
        , const int Cols
        , const int iVAlTresh
        , const int WndWidth
        , const int* d_pbin_metrics
        , const std::string & filename)*/ 

    
    std::string filename = "ouput.log";

    /*print_input_clusterization(thrust::raw_pointer_cast(d_fdmt_digitized.data())
        , rows
        , cols
        , *d_pTresh_digitized
        , WndWidth
        , d_pbin_metrics
        , filename
    );*/
    clusterization::clusterization_main(d_arr
        , rows
        , cols
        , *d_pTresh
        , WndWidth
        , d_pbin_metrics
        , filename
        );
  
    free(arr);
    free(norm); 


    return 0;
}

