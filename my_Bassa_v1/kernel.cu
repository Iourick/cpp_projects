#include "hip/hip_runtime.h"
﻿// ./kernel -n 8192 -N 32768 -P 001 -b 4 -d 0.5 1.0 80 -o OutImages -q L2012176_SAP000_B000_S0_P001_bf.h5
//.\my_Bassa_v1.exe -n 8192 -N 32768 -P 001 -b 4 -d 0.5 1.0 80 -o OutImages -q D:/BASSA/hdf5_data/L2012176_SAP000_B000_S0_P001_bf.h5
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <math.h>

#include <array>
#include <iostream>
#include <string>

#include <vector>
//#include <unistd.h>
#include <math.h>
#include <time.h>
#include<errno.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hdf5.h>
#include <stdlib.h>
#include "npy.hpp"
#include <vector>
#include <chrono>

#include <math_functions.h>
#include <complex>

#ifdef _WIN32
#include <direct.h>  // For Windows mkdir
#include <windows.h>
#define mkdir(path, mode) _mkdir(path)
#elif __linux__
#include <sys/stat.h>  // For Linux mkdir
#endif


#define HEADERSIZE 4096
#define DMCONSTANT 2.41e-10

#define _USE_MATH_DEFINES
#include <cmath>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

using namespace std;
// Struct for header information
struct header {
    int64_t headersize, buffersize;
    unsigned int nchan, nsamp, nbit, nif, nsub;
    int machine_id, telescope_id, nbeam, ibeam, sumif;
    double tstart, tsamp, fch1, foff, fcen, bwchan;
    double src_raj, src_dej, az_start, za_start;
    char source_name[80], ifstream[8], inpfile[80];
    char* rawfname[4];
};

struct header read_h5_header(char* fname);
void get_channel_chirp(double fcen, double bw, float dm, int nchan, int nbin, int nsub, hipfftComplex* c);
__global__ void transpose_unpadd_and_detect(hipfftComplex* cp1, hipfftComplex* cp2, int nbin, int nchan, int nfft, int nsub, int noverlap, int nsamp, float* fbuf);
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a, float s);
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b);
static __global__ void PointwiseComplexMultiply(hipfftComplex* a, hipfftComplex* b, hipfftComplex* c, int nx, int ny, int l, float scale);
__global__ void unpack_and_padd(char* dbuf0, char* dbuf1, char* dbuf2, char* dbuf3, int nsamp, int nbin, int nfft, int nsub, int noverlap, hipfftComplex* cp1, hipfftComplex* cp2);
__global__ void swap_spectrum_halves(hipfftComplex* cp1, hipfftComplex* cp2, int nx, int ny);
__global__ void compute_chirp(double fcen, double bw, float* dm, int nchan, int nbin, int nsub, int ndm, hipfftComplex* c);
__global__ void compute_block_sums(float* z, int nchan, int nblock, int nsum, float* bs1, float* bs2);
__global__ void compute_channel_statistics(int nchan, int nblock, int nsum, float* bs1, float* bs2, float* zavg, float* zstd);
__global__ void redigitize(float* z, int nchan, int nblock, int nsum, float* zavg, float* zstd, float zmin, float zmax, unsigned char* cz);
__global__ void decimate_and_redigitize(float* z, int ndec, int nchan, int nblock, int nsum, float* zavg, float* zstd, float zmin, float zmax, unsigned char* cz);
void write_filterbank_header(struct header h, FILE* file);
__global__ void kernel_Sum_cols(unsigned char* d_inp, const int NRows, const int NCols, int* d_arrOut);
int readHeader5InBinary(const char* strFilePath, header& Head);
int writeOptions(const char* strFilePath, const int noverlap, const int nbin, const int ndec, const char* obsid
    , const int part, const int device);
int readOptions(const char* strFilePath, int& noverlap, int& nbin, int& ndec, char* obsid
    , int& part, int& device);
__global__
void sum_rows_kernel(float* dfbuf, unsigned int  irows, unsigned int  icols, unsigned int numRowOut, float* d_parrImOut);

// Usage
void usage()
{
    printf("cdmt -P <part> -d <DM start,step,num> -D <GPU device> -b <ndec> -N <forward FFT size> -n <overlap region> -o <outputname> <file.h5>\n\n");
    printf("Compute coherently dedispersed SIGPROC filterbank files from LOFAR complex voltage data in HDF5 format.\n");
    printf("-P <part>        Specify part number for input file [integer, default: 0]\n");
    printf("-D <GPU device>  Select GPU device [integer, default: 0]\n");
    printf("-b <ndec>        Number of time samples to average [integer, default: 1]\n");
    printf("-d <DM start, step, num>  DM start and stepsize, number of DM trials\n");
    printf("-o <outputname>           Output filename [default: cdmt]\n");
    printf("-N <forward FFT size>     Forward FFT size [integer, default: 65536]\n");
    printf("-n <overlap region>       Overlap region [integer, default: 2048]\n");

    return;
}

int main(int argc, char* argv[])
{
    



    // creating directory for output images:
     // Specify the path for the new directory
    
    char *pathOutDir = NULL;     
    int i, nsamp, nfft, mbin, nvalid, nchan = 8, nbin = 65536, noverlap = 2048, nsub = 20, ndm, ndec = 1;
    int idm, iblock, nread, mchan, msamp, mblock, msum = 1024;
    char/** header,*/ * h5buf[4], * dh5buf[4];
    FILE* rawfile[4], * file;
    unsigned char* cbuf, * dcbuf;
    float* fbuf, * dfbuf;
    float* bs1, * bs2, * zavg, * zstd;
    hipfftComplex* cp1, * cp2, * dc, * cp1p, * cp2p;
    hipfftHandle ftc2cf, ftc2cb;
    int idist, odist, iembed, oembed, istride, ostride;
    dim3 blocksize, gridsize;
    struct header h5;
    clock_t startclock;
    float* dm, * ddm, dm_start, dm_step;
    char fname[128], fheader[1024], *h5fname = NULL, obsid[128] = "cdmt";
    int bytes_read;
    int part = 0, device = 0;
    int arg = 0;

    // default defining of command line parameters:
    char carr_ch[] = "D://BASSA//hdf5_data//L2012176_SAP000_B000_S0_P001_bf.h5";
    h5fname = carr_ch;
    char ch_tmp[] = "OutImages";
    pathOutDir = ch_tmp;
    dm_start = 0.5;
    dm_step = 1.0;
    ndm = 80;
    //noverlap = 8192;// 1<<13
    noverlap = 0;// 1<<13
    nbin = 32768;// 1<<15
    ndec = 4;
    part = 0;
    device = 0;
    //strcpy(obsid, "tmp/test");
    // ! default defining..

    if (argc > 1)
    {
        if (argc < 17)
        {
            std::cerr << "Usage: " << argv[0] << " -n <value> -N <value> -P <value> -b <value> -d <value1> <value2> <value3> -o <outputFile>  <inputFile>" << std::endl;
            return 1;
        }
        for (int i = 1; i < argc; ++i)
        {
            if (std::string(argv[i]) == "-n")
            {
                noverlap = std::atoi(argv[++i]);
                continue;
            }
            if (std::string(argv[i]) == "-P")
            {
                part = std::atoi(argv[++i]);
                continue;
            }
            if (std::string(argv[i]) == "-N")
            {
                nbin = std::atoi(argv[++i]);
                continue;
            }

            if (std::string(argv[i]) == "-b")
            {
                ndec = std::atoi(argv[++i]);
                continue;
            }
            if (std::string(argv[i]) == "-d")
            {
                // Split the comma-separated values
               // sscanf(argv[++i], "%f,%f,%f", &dm_start, &dm_step, &ndm);
                dm_start = std::atof(argv[++i]);
                dm_step = std::atof(argv[++i]);
                ndm = std::atof(argv[++i]);
                continue;
            }
            if (std::string(argv[i]) == "-o")
            {
                pathOutDir = argv[++i];
                continue;
            }
            if (std::string(argv[i]) == "-q")
            {
                h5fname  = argv[++i];
                continue;
            }         

           
            
        }
    }
    
    
    
#ifdef _WIN32
    _mkdir(pathOutDir);

#elif __linux__
    // Linux-specific code
    if (mkdir(pathOutDir, S_IRWXU) == 0) {
        std::cout << "Directory created successfully." << std::endl;
    }
    else {
        std::cerr << "Error creating directory." << std::endl;
    }
#else
    // Unsupported platform
    std::cerr << "Error: Unsupported platform" << std::endl;
#endif    
    //---------

    
    h5 = read_h5_header(h5fname);
    

    // Set number of subbands
    nsub = h5.nsub;

    // Adjust header for filterbank format
    h5.tsamp *= nchan * ndec;
    h5.nchan = nsub * nchan;
    h5.nbit = 8;
    h5.fch1 = h5.fcen + 0.5 * h5.nsub * h5.bwchan - 0.5 * h5.bwchan / nchan;
    h5.foff = -fabs(h5.bwchan / nchan);

    // Data size
    nvalid = nbin - 2 * noverlap;
    nsamp = 100 * nvalid;
    nfft = (int)ceil(nsamp / (float)nvalid);
    mbin = nbin / nchan;
    mchan = nsub * nchan;
    msamp = nsamp / nchan;
    mblock = msamp / msum;

    printf("nbin: %d nfft: %d nsub: %d mbin: %d nchan: %d nsamp: %d nvalid: %d\n", nbin, nfft, nsub, mbin, nchan, nsamp, nvalid);

    // Set device
    checkCudaErrors(hipSetDevice(device));

    // Allocate memory for complex timeseries
    checkCudaErrors(hipMalloc((void**)&cp1, sizeof(hipfftComplex) * nbin * nfft * nsub));
    checkCudaErrors(hipMalloc((void**)&cp2, sizeof(hipfftComplex) * nbin * nfft * nsub));
    checkCudaErrors(hipMalloc((void**)&cp1p, sizeof(hipfftComplex) * nbin * nfft * nsub));
    checkCudaErrors(hipMalloc((void**)&cp2p, sizeof(hipfftComplex) * nbin * nfft * nsub));

    // Allocate device memory for chirp
    std::cout << "nbin = " << nbin << "  nsub = " << nsub << "  ndm = " << ndm << std::endl;
    checkCudaErrors(hipMalloc((void**)&dc, sizeof(hipfftComplex) * nbin * nsub * ndm));

    // Allocate device memory for block sums
    checkCudaErrors(hipMalloc((void**)&bs1, sizeof(float) * mblock * mchan));
    checkCudaErrors(hipMalloc((void**)&bs2, sizeof(float) * mblock * mchan));

    // Allocate device memory for channel averages and standard deviations
    checkCudaErrors(hipMalloc((void**)&zavg, sizeof(float) * mchan));
    checkCudaErrors(hipMalloc((void**)&zstd, sizeof(float) * mchan));

   
    for (i = 0; i < 4; i++) {
        h5buf[i] = (char*)malloc(sizeof(char) * nsamp * nsub);
        checkCudaErrors(hipMalloc((void**)&dh5buf[i], sizeof(char) * nsamp * nsub));
    }

    // Allocate output buffers
    fbuf = (float*)malloc(sizeof(float) * nsamp * nsub);
    checkCudaErrors(hipMalloc((void**)&dfbuf, sizeof(float) * nsamp * nsub));
    cbuf = (unsigned char*)malloc(sizeof(unsigned char) * msamp * mchan / ndec);
    checkCudaErrors(hipMalloc((void**)&dcbuf, sizeof(unsigned char) * msamp * mchan / ndec));

    // Allocate DMs and copy to device
    dm = (float*)malloc(sizeof(float) * ndm);
    for (idm = 0; idm < ndm; idm++)
        dm[idm] = dm_start + (float)idm * dm_step;
    checkCudaErrors(hipMalloc((void**)&ddm, sizeof(float) * ndm));
    checkCudaErrors(hipMemcpy(ddm, dm, sizeof(float) * ndm, hipMemcpyHostToDevice));

    // Generate FFT plan (batch in-place forward FFT)
    idist = nbin;  odist = nbin;  iembed = nbin;  oembed = nbin;  istride = 1;  ostride = 1; 

    checkCudaErrors(hipfftPlanMany(&ftc2cf, 1, &nbin, &iembed, istride, idist, &oembed, ostride, odist, HIPFFT_C2C, nfft * nsub));
    //hipfftPlanMany(&ftc2cf,            // Plan handle
    //    1,                   // Rank (number of dimensions in the transform)
    //    &nbin,               // Dimensions of the transform =32768 = 1<<15
    //    &iembed,             // Input size and embed dimension (can be NULL) =32768 
    //    istride,             // Input strides (distance between successive input elements) = 1
    //    idist,               // Input distance (distance between
    //                          the first element of successive batches) =32768
    //    &oembed,             // Output size and embed dimension (can be NULL) =32768
    //    ostride,             // Output strides (distance between successive output elements) = 1
    //    odist,               // Output distance (distance between the first element
    //                           of successive batches)    =32768
    //    HIPFFT_C2C,           // Type of transform (in this case, complex to complex)
    //    nfft * nsub);        // Number of FFTs (batch size) = 100 * 20

    // Generate FFT plan (batch in-place backward FFT)
    idist = mbin;  odist = mbin;  iembed = mbin;  oembed = mbin;  istride = 1;  ostride = 1;
    checkCudaErrors(hipfftPlanMany(&ftc2cb, 1, &mbin, &iembed, istride, idist, &oembed, ostride, odist, HIPFFT_C2C, nchan * nfft * nsub));


    hipDeviceSynchronize();
    // Compute chirp
    blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
    gridsize.x = nsub / blocksize.x + 1; gridsize.y = nchan / blocksize.y + 1; gridsize.z = ndm / blocksize.z + 1;
    double bw = ((double)(nsub)) * h5.bwchan;
    compute_chirp << <gridsize, blocksize >> > (h5.fcen, bw, ddm, nchan, nbin, nsub, ndm, dc);

   // (double fcen, double bw, float* dm, int nchan, int nbin, int nsub, int ndm, hipfftComplex * c)
    hipDeviceSynchronize();
    std::vector<std::complex<float>> data4(nbin* nsub* ndm, 0);
    hipMemcpy(data4.data(), dc, nbin* nsub* ndm * sizeof(std::complex<float>),
        hipMemcpyDeviceToHost);
    // Read files
   for (i = 0; i < 4; i++) {
        rawfile[i] = fopen(h5.rawfname[i], "rb");
        
    }

    // Loop over input file contents
    // memory allocation for input image
    float* d_parrImOut = NULL;
    unsigned int irows = h5.nchan;
    unsigned int icols = nsamp / nchan;
    checkCudaErrors(hipMallocManaged((void**)&d_parrImOut, sizeof(float) * irows * icols));

    long long totalBytes = 0;
    long long totalLength = 0;
    for (iblock = 0;; iblock++)
    {
        // Read block
        startclock = clock();
        for (i = 0; i < 4; i++)
        {
            nread = fread(h5buf[i], sizeof(char), nsamp * nsub, rawfile[i]);      
            if (nread == 0)
                break;
            totalBytes += nread;
            nread = nread / nsub;
            totalLength += nread;
        }
        if (nread == 0)
        {
            break;
        }
        printf("Block: %d: Read %d MB in %.2f s\n", iblock, sizeof(char) * nread * nsub * 4 / (1 << 20), (float)(clock() - startclock) / CLOCKS_PER_SEC);

        // Copy buffers to device
        startclock = clock();
        for (i = 0; i < 4; i++)
        {
            checkCudaErrors(hipMemcpy(dh5buf[i], h5buf[i], sizeof(char) * nread * nsub, hipMemcpyHostToDevice));
        }
        // Unpack data and padd data
        blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
        gridsize.x = nbin / blocksize.x + 1; gridsize.y = nfft / blocksize.y + 1; gridsize.z = nsub / blocksize.z + 1;
        unpack_and_padd << <gridsize, blocksize >> > (dh5buf[0], dh5buf[1], dh5buf[2], dh5buf[3], nread, nbin, nfft, nsub, noverlap, cp1p, cp2p);
        
        
        std::vector<std::complex<float>> data2(irows * icols, 0);
        hipMemcpy(data2.data(), cp1p, irows *icols * sizeof(std::complex<float>),
            hipMemcpyDeviceToHost);
        

        // Perform FFTs
        checkCudaErrors(hipfftExecC2C(ftc2cf, (hipfftComplex*)cp1p, (hipfftComplex*)cp1p, HIPFFT_FORWARD));
        checkCudaErrors(hipfftExecC2C(ftc2cf, (hipfftComplex*)cp2p, (hipfftComplex*)cp2p, HIPFFT_FORWARD));

        std::vector<std::complex<float>> data3(irows* icols, 0);
        hipMemcpy(data3.data(), cp1p, irows* icols * sizeof(std::complex<float>),
            hipMemcpyDeviceToHost);

        // Swap spectrum halves for large FFTs
        blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
        gridsize.x = nbin / blocksize.x + 1; gridsize.y = nfft * nsub / blocksize.y + 1; gridsize.z = 1;
        swap_spectrum_halves << <gridsize, blocksize >> > (cp1p, cp2p, nbin, nfft * nsub);

        // Loop over dms
        for (idm = 0; idm < ndm; idm++) {

            // Perform complex multiplication of FFT'ed data with chirp
            blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
            gridsize.x = nbin * nsub / blocksize.x + 1; gridsize.y = nfft / blocksize.y + 1; gridsize.z = 1;
            PointwiseComplexMultiply << <gridsize, blocksize >> > (cp1p, dc, cp1, nbin * nsub, nfft, idm, 1.0 / (float)nbin);
            PointwiseComplexMultiply << <gridsize, blocksize >> > (cp2p, dc, cp2, nbin * nsub, nfft, idm, 1.0 / (float)nbin);
            hipMemcpy(data3.data(), cp1, irows * icols * sizeof(std::complex<float>),
                hipMemcpyDeviceToHost);
            // Swap spectrum halves for small FFTs
            blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
            gridsize.x = mbin / blocksize.x + 1; gridsize.y = nchan * nfft * nsub / blocksize.y + 1; gridsize.z = 1;
            swap_spectrum_halves << <gridsize, blocksize >> > (cp1, cp2, mbin, nchan * nfft * nsub);

            // Perform FFTs
            checkCudaErrors(hipfftExecC2C(ftc2cb, (hipfftComplex*)cp1, (hipfftComplex*)cp1, HIPFFT_BACKWARD));
            checkCudaErrors(hipfftExecC2C(ftc2cb, (hipfftComplex*)cp2, (hipfftComplex*)cp2, HIPFFT_BACKWARD));

            // Detect data
            blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
            gridsize.x = mbin / blocksize.x + 1; gridsize.y = nchan / blocksize.y + 1; gridsize.z = nfft / blocksize.z + 1;
            transpose_unpadd_and_detect << <gridsize, blocksize >> > (cp1, cp2, mbin, nchan, nfft, nsub, noverlap / nchan, nread / nchan, dfbuf);
            hipDeviceSynchronize();
            

            float* arrt = (float*)malloc(irows * icols * sizeof(float));
            hipMemcpy(arrt, dfbuf, irows * icols * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            free(arrt);

            blocksize.x = 1024; blocksize.y = 1; blocksize.z = 1;
            gridsize.x = (icols + blocksize.x -1) / blocksize.x; gridsize.y =  1; gridsize.z =  1;
            sum_rows_kernel << <gridsize, blocksize >> > (dfbuf, irows, icols, idm, d_parrImOut);
            hipDeviceSynchronize();
            


            
        }
        hipDeviceSynchronize();
        std::array<long unsigned, 2> leshape{irows, icols };

        char strFilePath[300] = { 0 };        
        // Use snprintf to format the entire string in one operation
        char chstr[20] = { 0 };
        sprintf(chstr, "/image_%d.npy", iblock);

        strcpy(strFilePath, pathOutDir);
        strcat(strFilePath, chstr);
      
        npy::SaveArrayAsNumpy(strFilePath, false, leshape.size(), leshape.data(), d_parrImOut);

        printf("Processed %d DMs in %.2f s\n", ndm, (float)(clock() - startclock) / CLOCKS_PER_SEC);
    }

        hipFree(d_parrImOut);
        std::cout << "totalBytes = " << totalBytes/4 << "  totalLength = " << totalLength/4 << std::endl;
        

    // Close files
    for (i = 0; i < 4; i++)
        fclose(rawfile[i]);

    for (i = 0; i < 4; i++) {
        free(h5buf[i]);
        hipFree(dh5buf);
        free(h5.rawfname[i]);
    }
    free(fbuf);
    free(dm);
    free(cbuf);
    //free(outfile);

    hipFree(dfbuf);
    hipFree(dcbuf);
    hipFree(cp1);
    hipFree(cp2);
    hipFree(cp1p);
    hipFree(cp2p);
    hipFree(dc);
    hipFree(bs1);
    hipFree(bs2);
    hipFree(zavg);
    hipFree(zstd);
    hipFree(ddm);

    // Free plan
    hipfftDestroy(ftc2cf);
    hipfftDestroy(ftc2cb);

    return 0;
}
// !!!!!!
// This is a simple H5 reader for complex voltage data. Very little
// error checking is done.
struct header read_h5_header(char* fname)
{
    int i, len, ibeam, isap;
    struct header h;
    hid_t file_id, attr_id, sap_id, beam_id, memtype, group_id, space, coord_id;
    char* string, * pch;
    const char* stokes[] = { "_S0_","_S1_","_S2_","_S3_" };
    char* froot, * fpart, * ftest, group[32];
    FILE* file;

    // Find filenames
    for (i = 0; i < 4; i++) {
        pch = strstr(fname, stokes[i]);
        if (pch != NULL)
            break;
    }
    len = strlen(fname) - strlen(pch);
    froot = (char*)malloc(sizeof(char) * (len + 1));
    memset(froot, 0, sizeof(char) * (len + 1));
    fpart = (char*)malloc(sizeof(char) * (strlen(pch) - 6));
    memset(fpart, 0, sizeof(char) * (strlen(pch) - 6));
    ftest = (char*)malloc(sizeof(char) * (len + 20));
    memset(ftest,0, sizeof(char) * (len + 20));
    strncpy(froot, fname, len);
    strncpy(fpart, pch + 4, strlen(pch) - 7);

    // Check files
    for (i = 0; i < 4; i++) {
        // Format file name
        sprintf(ftest, "%s_S%d_%s.raw", froot, i, fpart);
        // Try to open
        if ((file = fopen(ftest, "r")) != NULL) {
            fclose(file);
        }
        else {
            fprintf(stderr, "Raw file %s not found\n", ftest);
            exit(-1);
        }
        h.rawfname[i] = (char*)malloc(sizeof(char) * (strlen(ftest) + 1));
        strcpy(h.rawfname[i], ftest);
    }

    // Get beam number
    for (i = 0; i < 4; i++) {
        pch = strstr(fname, "_B");
        if (pch != NULL)
            break;
    }
    sscanf(pch + 2, "%d", &ibeam);

    // Get SAP number
    for (i = 0; i < 4; i++) {
        pch = strstr(fname, "_SAP");
        if (pch != NULL)
            break;
    }
    sscanf(pch + 4, "%d", &isap);

    // Free
    free(froot);
    free(fpart);
    free(ftest);

    // Open file
    file_id = H5Fopen(fname, H5F_ACC_RDONLY, H5P_DEFAULT);

    // Open subarray pointing group
    sprintf(group, "SUB_ARRAY_POINTING_%03d", isap);
    sap_id = H5Gopen(file_id, group, H5P_DEFAULT);

    // Start MJD
    attr_id = H5Aopen(sap_id, "EXPTIME_START_MJD", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.tstart);
    H5Aclose(attr_id);

    // Declination
    attr_id = H5Aopen(sap_id, "POINT_DEC", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.src_dej);
    H5Aclose(attr_id);

    // Right ascension
    attr_id = H5Aopen(sap_id, "POINT_RA", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.src_raj);
    H5Aclose(attr_id);

    // Open beam
    sprintf(group, "BEAM_%03d", ibeam);
    beam_id = H5Gopen(sap_id, group, H5P_DEFAULT);

    // Number of samples
    attr_id = H5Aopen(beam_id, "NOF_SAMPLES", H5P_DEFAULT);
    H5Aread(attr_id, H5T_STD_U32LE, &h.nsamp);    
    H5Aclose(attr_id);

    // Center frequency
    attr_id = H5Aopen(beam_id, "BEAM_FREQUENCY_CENTER", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.fcen);
    H5Aclose(attr_id);

    // Center frequency unit
    attr_id = H5Aopen(beam_id, "BEAM_FREQUENCY_CENTER_UNIT", H5P_DEFAULT);
    memtype = H5Tcopy(H5T_C_S1);
    H5Tset_size(memtype, H5T_VARIABLE);
    H5Aread(attr_id, memtype, &string);
    H5Aclose(attr_id);
    if (strcmp(string, "Hz") == 0)
        h.fcen /= 1e6;

    // Channel bandwidth
    attr_id = H5Aopen(beam_id, "CHANNEL_WIDTH", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.bwchan);
    H5Aclose(attr_id);

    // Center frequency unit
    attr_id = H5Aopen(beam_id, "CHANNEL_WIDTH_UNIT", H5P_DEFAULT);
    memtype = H5Tcopy(H5T_C_S1);
    H5Tset_size(memtype, H5T_VARIABLE);
    H5Aread(attr_id, memtype, &string);
    H5Aclose(attr_id);
    if (strcmp(string, "Hz") == 0)
        h.bwchan /= 1e6;

    // Get source
    attr_id = H5Aopen(beam_id, "TARGETS", H5P_DEFAULT);
    memtype = H5Tcopy(H5T_C_S1);
    H5Tset_size(memtype, H5T_VARIABLE);
    H5Aread(attr_id, memtype, &string);
    H5Aclose(attr_id);
    strcpy(h.source_name, string);

    // Open coordinates
    coord_id = H5Gopen(beam_id, "COORDINATES", H5P_DEFAULT);

    // Open coordinate 0
    group_id = H5Gopen(coord_id, "COORDINATE_0", H5P_DEFAULT);

    // Sampling time
    attr_id = H5Aopen(group_id, "INCREMENT", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.tsamp);
    H5Aclose(attr_id);

    // Close group
    H5Gclose(group_id);

    // Open coordinate 1
    group_id = H5Gopen(coord_id, "COORDINATE_1", H5P_DEFAULT);

    // Number of subbands
    attr_id = H5Aopen(group_id, "AXIS_VALUES_WORLD", H5P_DEFAULT);
    space = H5Aget_space(attr_id);
    h.nsub = H5Sget_simple_extent_npoints(space);
    H5Aclose(attr_id);

    // Close group
    H5Gclose(group_id);

    // Close coordinates
    H5Gclose(coord_id);

    // Close beam, sap and file
    H5Gclose(beam_id);
    H5Gclose(sap_id);
    H5Fclose(file_id);

    return h;
}

// Scale hipfftComplex 
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a, float s)
{
    hipfftComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Pointwise complex multiplication (and scaling)
static __global__ void PointwiseComplexMultiply(hipfftComplex* a, hipfftComplex* b, hipfftComplex* c, int nx, int ny, int l, float scale)
{
    int i, j, k;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        k = i + nx * j;
        c[k] = ComplexScale(ComplexMul(a[k], b[i + nx * l]), scale);
    }
}

// Compute chirp
__global__ void compute_chirp(double fcen, double bw, float* dm, int nchan, int nbin, int nsub, int ndm, hipfftComplex* c)
{
  // printf("fcen = %f ; bw = %f ; nchan = %i ; /*nbin = %i ; nsub =  %i ; ndm = %i*/\n", fcen, bw, nchan/*, nbin, nsub, ndm*/);
  printf("fcen = %f ; bw = %f \n", fcen, bw);
    //int ibin, ichan, isub, idm, mbin, idx;
    //double s, rt, t, f, fsub, fchan, bwchan, bwsub;

    //// Number of channels per subband
    //mbin = nbin / nchan;

    //// Subband bandwidth
    //bwsub = bw / nsub;

    //// Channel bandwidth
    //bwchan = bw / (nchan * nsub);

    //// Indices of input data
    //isub = blockIdx.x * blockDim.x + threadIdx.x;
    //ichan = blockIdx.y * blockDim.y + threadIdx.y;
    //idm = blockIdx.z * blockDim.z + threadIdx.z;

    //// Keep in range
    //if (isub < nsub && ichan < nchan && idm < ndm)
    //  {
    //    // Main constant
    //    s = 2.0 * M_PI * dm[idm] / DMCONSTANT;
    //    printf("s = %f  \n",s);
    //    // Frequencies
    //    fsub = fcen - 0.5 * bw + bw * (float)isub / (float)nsub + 0.5 * bw / (float)nsub;
    //    fchan = fsub - 0.5 * bwsub + bwsub * (float)ichan / (float)nchan + 0.5 * bwsub / (float)nchan;

    //    // Loop over bins in channel
    //    for (ibin = 0; ibin < mbin; ibin++)
    //    {
    //        // Bin frequency
    //        f = -0.5 * bwchan + bwchan * (float)ibin / (float)mbin + 0.5 * bwchan / (float)mbin;

    //        //// Phase delay
    //        if (isub == 1)
    //        {
    //           // rt = s;// -f * f * s / ((fchan + f) * fchan * fchan);
    //            //printf("f = %f; s = %f ; fchan= %f \n", f, s, fchan);
    //        }
    //       // rt = -f * f * s / ((fchan + f) * fchan * fchan);

    //        //// Taper
    //        //t = 1.0 / sqrt(1.0 + pow((f / (0.47 * bwchan)), 80));

    //        //// Index
    //        //idx = ibin + ichan * mbin + isub * mbin * nchan + idm * nsub * mbin * nchan;

    //        //// Chirp
    //        //c[idx].x = cos(rt) * t;
    //        //c[idx].y = sin(rt) * t;
    //        //printf("t = %f ; rt = %f ; ", t, rt);
    //    }
    //}

   // return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution.
__global__ void unpack_and_padd(char* dbuf0, char* dbuf1, char* dbuf2, char* dbuf3, int nsamp, int nbin, int nfft, int nsub, int noverlap, hipfftComplex* cp1, hipfftComplex* cp2)
{
    int64_t ibin, ifft, isamp, isub, idx1, idx2;

    // Indices of input data
    ibin = blockIdx.x * blockDim.x + threadIdx.x;
    ifft = blockIdx.y * blockDim.y + threadIdx.y;
    isub = blockIdx.z * blockDim.z + threadIdx.z;

    // Only compute valid threads
    if (ibin < nbin && ifft < nfft && isub < nsub) {
        idx1 = ibin + nbin * isub + nsub * nbin * ifft;
        isamp = ibin + (nbin - 2 * noverlap) * ifft - noverlap;
        idx2 = isub + nsub * isamp;
        if (isamp < 0 || isamp >= nsamp) {
            cp1[idx1].x = 0.0;
            cp1[idx1].y = 0.0;
            cp2[idx1].x = 0.0;
            cp2[idx1].y = 0.0;
        }
        else {
            cp1[idx1].x = (float)dbuf0[idx2];
            cp1[idx1].y = (float)dbuf1[idx2];
            cp2[idx1].x = (float)dbuf2[idx2];
            cp2[idx1].y = (float)dbuf3[idx2];
        }
    }

    return;
}

// Since complex-to-complex FFTs put the center frequency at bin zero
// in the frequency domain, the two halves of the spectrum need to be
// swapped.
__global__ void swap_spectrum_halves(hipfftComplex* cp1, hipfftComplex* cp2, int nx, int ny)
{
    int64_t i, j, k, l, m;
    hipfftComplex tp1, tp2;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx / 2 && j < ny) {
        if (i < nx / 2)
            k = i + nx / 2;
        else
            k = i - nx / 2;
        l = i + nx * j;
        m = k + nx * j;
        tp1.x = cp1[l].x;
        tp1.y = cp1[l].y;
        tp2.x = cp2[l].x;
        tp2.y = cp2[l].y;
        cp1[l].x = cp1[m].x;
        cp1[l].y = cp1[m].y;
        cp2[l].x = cp2[m].x;
        cp2[l].y = cp2[m].y;
        cp1[m].x = tp1.x;
        cp1[m].y = tp1.y;
        cp2[m].x = tp2.x;
        cp2[m].y = tp2.y;
    }

    return;
}

// After the segmented FFT the data is in a cube of nbin by nchan by
// nfft, where nbin and nfft are the time indices. Here we rearrange
// the 3D data cube into a 2D array of frequency and time, while also
// removing the overlap regions and detecting (generating Stokes I).
__global__ void transpose_unpadd_and_detect(hipfftComplex* cp1, hipfftComplex* cp2, int nbin, int nchan, int nfft, int nsub, int noverlap, int nsamp, float* fbuf)
{
    int64_t ibin, ichan, ifft, isub, isamp, idx1, idx2;

    ibin = blockIdx.x * blockDim.x + threadIdx.x;
    ichan = blockIdx.y * blockDim.y + threadIdx.y;
    ifft = blockIdx.z * blockDim.z + threadIdx.z;
    if (ibin < nbin && ichan < nchan && ifft < nfft) {
        // Loop over subbands
        for (isub = 0; isub < nsub; isub++) {
            // Padded array index
            //      idx1=ibin+nbin*isub+nsub*nbin*(ichan+nchan*ifft);
            idx1 = ibin + ichan * nbin + (nsub - isub - 1) * nbin * nchan + ifft * nbin * nchan * nsub;

            // Time index
            isamp = ibin + (nbin - 2 * noverlap) * ifft - noverlap;

            // Output array index
            idx2 = (nchan - ichan - 1) + isub * nchan + nsub * nchan * isamp;

            // Select data points from valid region
            if (ibin >= noverlap && ibin <= nbin - noverlap && isamp >= 0 && isamp < nsamp)
                fbuf[idx2] = cp1[idx1].x * cp1[idx1].x + cp1[idx1].y * cp1[idx1].y + cp2[idx1].x * cp2[idx1].x + cp2[idx1].y * cp2[idx1].y;
        }
    }

    return;
}

void send_string(const char* string, FILE* file)
{
    int len;

    len = strlen(string);
    fwrite(&len, sizeof(int), 1, file);
    fwrite(string, sizeof(char), len, file);

    return;
}

void send_float(const char* string, float x, FILE* file)
{
    send_string(string, file);
    fwrite(&x, sizeof(float), 1, file);

    return;
}

void send_int(const char* string, int x, FILE* file)
{
    send_string(string, file);
    fwrite(&x, sizeof(int), 1, file);

    return;
}

void send_double(const char* string, double x, FILE* file)
{
    send_string(string, file);
    fwrite(&x, sizeof(double), 1, file);

    return;
}

double dec2sex(double x)
{
    double d, sec, min, deg;
    char sign;
    char tmp[32];

    sign = (x < 0 ? '-' : ' ');
    x = 3600.0 * fabs(x);

    sec = fmod(x, 60.0);
    x = (x - sec) / 60.0;
    min = fmod(x, 60.0);
    x = (x - min) / 60.0;
    deg = x;

    sprintf(tmp, "%c%02d%02d%09.6lf", sign, (int)deg, (int)min, sec);
    sscanf(tmp, "%lf", &d);

    return d;
}

void write_filterbank_header(struct header h, FILE* file)
{
    double ra, de;


    ra = dec2sex(h.src_raj / 15.0);
    de = dec2sex(h.src_dej);

    send_string("HEADER_START", file);
    send_string("rawdatafile", file);
    send_string(h.rawfname[0], file);
    send_string("source_name", file);
    send_string(h.source_name, file);
    send_int("machine_id", 11, file);
    send_int("telescope_id", 11, file);
    send_double("src_raj", ra, file);
    send_double("src_dej", de, file);
    send_int("data_type", 1, file);
    send_double("fch1", h.fch1, file);
    send_double("foff", h.foff, file);
    send_int("nchans", h.nchan, file);
    send_int("nbeams", 0, file);
    send_int("ibeam", 0, file);
    send_int("nbits", h.nbit, file);
    send_double("tstart", h.tstart, file);
    send_double("tsamp", h.tsamp, file);
    send_int("nifs", 1, file);
    send_string("HEADER_END", file);

    return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_block_sums(float* z, int nchan, int nblock, int nsum, float* bs1, float* bs2)
{
    int64_t ichan, iblock, isum, idx1, idx2;

    ichan = blockIdx.x * blockDim.x + threadIdx.x;
    iblock = blockIdx.y * blockDim.y + threadIdx.y;
    if (ichan < nchan && iblock < nblock) {
        idx1 = ichan + nchan * iblock;
        bs1[idx1] = 0.0;
        bs2[idx1] = 0.0;
        for (isum = 0; isum < nsum; isum++) {
            idx2 = ichan + nchan * (isum + iblock * nsum);
            bs1[idx1] += z[idx2];
            bs2[idx1] += z[idx2] * z[idx2];
        }
    }

    return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_channel_statistics(int nchan, int nblock, int nsum, float* bs1, float* bs2, float* zavg, float* zstd)
{
    int64_t ichan, iblock, idx1;
    double s1, s2;

    ichan = blockIdx.x * blockDim.x + threadIdx.x;
    if (ichan < nchan) {
        s1 = 0.0;
        s2 = 0.0;
        for (iblock = 0; iblock < nblock; iblock++) {
            idx1 = ichan + nchan * iblock;
            s1 += bs1[idx1];
            s2 += bs2[idx1];
        }
        zavg[ichan] = s1 / (float)(nblock * nsum);
        zstd[ichan] = s2 / (float)(nblock * nsum) - zavg[ichan] * zavg[ichan];
        zstd[ichan] = sqrt(zstd[ichan]);
    }

    return;
}

// Redigitize the filterbank to 8 bits in segments
__global__ void redigitize(float* z, int nchan, int nblock, int nsum, float* zavg, float* zstd, float zmin, float zmax, unsigned char* cz)
{
    int64_t ichan, iblock, isum, idx1;
    float zoffset, zscale;

    ichan = blockIdx.x * blockDim.x + threadIdx.x;
    iblock = blockIdx.y * blockDim.y + threadIdx.y;
    if (ichan < nchan && iblock < nblock) {
        zoffset = zavg[ichan] - zmin * zstd[ichan];
        zscale = (zmin + zmax) * zstd[ichan];

        for (isum = 0; isum < nsum; isum++) {
            idx1 = ichan + nchan * (isum + iblock * nsum);
            z[idx1] -= zoffset;
            z[idx1] *= 256.0 / zscale;
            cz[idx1] = (unsigned char)z[idx1];
            if (z[idx1] < 0.0) cz[idx1] = 0;
            if (z[idx1] > 255.0) cz[idx1] = 255;
        }
    }

    return;
}

// Decimate and Redigitize the filterbank to 8 bits in segments
__global__ void decimate_and_redigitize(float* z, int ndec, int nchan, int nblock, int nsum, float* zavg, float* zstd, float zmin, float zmax, unsigned char* cz)
{
    int64_t ichan, iblock, isum, idx1, idx2, idec;
    float zoffset, zscale, ztmp;

    ichan = blockIdx.x * blockDim.x + threadIdx.x;
    iblock = blockIdx.y * blockDim.y + threadIdx.y;
    if (ichan < nchan && iblock < nblock) {
        zoffset = zavg[ichan] - zmin * zstd[ichan];
        zscale = (zmin + zmax) * zstd[ichan];

        for (isum = 0; isum < nsum; isum += ndec) {
            idx2 = ichan + nchan * (isum / ndec + iblock * nsum / ndec);
            for (idec = 0, ztmp = 0.0; idec < ndec; idec++) {
                idx1 = ichan + nchan * (isum + idec + iblock * nsum);
                ztmp += z[idx1];
            }
            ztmp /= (float)ndec;
            ztmp -= zoffset;
            ztmp *= 256.0 / zscale;
            cz[idx2] = (unsigned char)ztmp;
            if (ztmp < 0.0) cz[idx2] = 0;
            if (ztmp > 255.0) cz[idx2] = 255;
        }
    }

    return;
}

//-----------------------------------------------------------------
__global__ void kernel_Sum_cols(unsigned char* d_inp, const int NRows, const int NCols, int* d_arrOut)
{
    extern __shared__ int sdata[];

    unsigned char* pTemp = d_inp + NCols * blockIdx.x + threadIdx.x;

    unsigned int tid = threadIdx.x;
    unsigned int i = tid;
    if (tid >= NCols)
    {
        sdata[tid] = 0;

    }
    else
    {
        int localSum = 0;



        while (i < NCols)
        {
            localSum += (int)(pTemp[i]);
            pTemp += blockDim.x;
            i += blockDim.x;

        }


    }
    __syncthreads();

    // Parallel reduction within the block to sum partial sums
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }

    // Only thread 0 within each block computes the block's sum
    if (tid == 0)
    {
        d_arrOut[blockIdx.x] = sdata[0];

    }
    __syncthreads();

}
//---------------------------------------------------
int readHeader5InBinary(const char* strFilePath, header& Head)
{

    // Open file for reading in binary mode
    FILE* file = fopen(strFilePath, "rb");

    if (file == NULL) {
        perror("Error opening file");
        return 1;
    }



    // Write the variables to the file

    fread(&Head.headersize, sizeof(int64_t), 1, file);
    fread(&Head.buffersize, sizeof(int64_t), 1, file);//
    fread(&Head.nchan, sizeof(unsigned int), 1, file);
    fread(&Head.nsamp, sizeof(unsigned int), 1, file);
    fread(&Head.nbit, sizeof(unsigned int), 1, file);
    fread(&Head.nif, sizeof(unsigned int), 1, file);
    fread(&Head.nsub, sizeof(unsigned int), 1, file);


    fread(&Head.machine_id, sizeof(int), 1, file);
    fread(&Head.telescope_id, sizeof(int), 1, file);
    fread(&Head.nbeam, sizeof(int), 1, file);
    fread(&Head.ibeam, sizeof(int), 1, file);
    fread(&Head.sumif, sizeof(int), 1, file);


    fread(&Head.tstart, sizeof(double), 1, file);
    fread(&Head.tsamp, sizeof(double), 1, file);
    fread(&Head.fch1, sizeof(double), 1, file);
    fread(&Head.foff, sizeof(double), 1, file);
    fread(&Head.fcen, sizeof(double), 1, file);
    fread(&Head.bwchan, sizeof(double), 1, file);


    fread(&Head.src_raj, sizeof(double), 1, file);
    fread(&Head.src_dej, sizeof(double), 1, file);
    fread(&Head.az_start, sizeof(double), 1, file);
    fread(&Head.za_start, sizeof(double), 1, file);




    fread(Head.source_name, sizeof(char), 80, file);
    fread(Head.ifstream, sizeof(char), 8, file);
    fread(Head.inpfile, sizeof(char), 8, file);


    for (int i = 0; i < 4; ++i)
    {
        int len = 0;
        fread(&len, sizeof(int), 1, file);
        Head.rawfname[i] = new char[len + 1];
        memset(Head.rawfname[i], 0, (1 + len) * sizeof(char));

        fread(Head.rawfname[i], sizeof(char), len, file);

    }

    // Close the file
    fclose(file);

    return 0;
}
//--------------------------------------------------
int writeOptions(const char* strFilePath, const int noverlap, const int nbin, const int ndec, const char* obsid
    , const int part, const int device)
{
    FILE* file = fopen(strFilePath, "wb");

    if (file == NULL) {
        perror("Error opening file");
        return 1;
    }
    // Write the variables to the file
    fwrite(&noverlap, sizeof(int), 1, file);
    fwrite(&nbin, sizeof(int), 1, file);//
    fwrite(&ndec, sizeof(int), 1, file);//
    int len = strlen(obsid);
    fwrite(&len, sizeof(int), 1, file);
    fwrite(obsid, sizeof(char), len, file);
    fwrite(&part, sizeof(int), 1, file);
    fwrite(&device, sizeof(int), 1, file);
    fclose(file);
    return 0;

}
//-------------------------------------

int readOptions(const char* strFilePath, int& noverlap, int& nbin, int& ndec, char* obsid
    , int& part, int& device)
{
    // Open file for reading in binary mode
    FILE* file = fopen(strFilePath, "rb");

    if (file == NULL) {
        perror("Error opening file");
        return 1;
    }
    // Write the variables to the file
    fread(&noverlap, sizeof(int), 1, file);
    fread(&nbin, sizeof(int), 1, file);//
    //fread(&ndec, sizeof(int), 1, file);//
    ndec = 4;
    int len = 0;
    fread(&len, sizeof(int), 1, file);
    fread(obsid, sizeof(char), len, file);
    fread(&part, sizeof(int), 1, file);
    fread(&device, sizeof(int), 1, file);


    // Close the file
    fclose(file);

    return 0;
}
//-----------------------------------------------------------
__global__
void sum_rows_kernel(float* dfbuf, unsigned int  irows, unsigned int  icols, unsigned int numRowOut, float* d_parrImOut)
{
    const int ind = threadIdx.x + blockIdx.x* blockDim.x ;
    
    if (ind >= icols)
    {
        return;
    }
    float sum = 0;
    float* p = &dfbuf[ind];
    for (int i = 0; i < irows; ++i)
    {
        sum += *p;
        p += icols;
    }
    d_parrImOut[icols * numRowOut + ind] = sum;
}