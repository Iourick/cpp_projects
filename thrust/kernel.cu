﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <iostream>
struct structTemp {
    int ma;
    int md;
};
int main()
{
    std::vector<std::vector<int>> coordinates;

    thrust::device_vector<structTemp > d_vec_struct(10);
    
    // Initialize a device vector with 10 elements
    thrust::device_vector<int> d_vec(10);
    thrust::device_vector<int> d_vec1(1);
    d_vec1 = d_vec1;
    // Fill the vector with some values
    for (int i = 0; i < d_vec.size(); i++) {
        d_vec[i] = 10 - i;
    }
    d_vec1 = d_vec;
    int ia = d_vec1[0];
    // Sort the vector in ascending order
    thrust::sort(d_vec.begin(), d_vec.end());

    // Print the sorted values
    std::cout << "Sorted values: ";
    for (int i = 0; i < d_vec.size(); i++) {
        std::cout << d_vec[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
