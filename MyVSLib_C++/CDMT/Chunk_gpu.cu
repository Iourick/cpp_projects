#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Chunk_gpu.cuh"
#include "yr_cart.h"
#include <vector>
#include "OutChunkHeader.h"

#include "Constants.h"


#include <chrono>

#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include <math_functions.h>
#include "aux_kernels.cuh"
#include "Detection.cuh"
#include "Cleaning.cuh"
#include <complex>

#include "Fragment.h"
#include "npy.hpp"
#include "TelescopeHeader.h"



//  
//#ifdef _WIN32 // Windows
//
//#include <Windows.h>
//
//    void emitSound(int frequency, int duration) {
//        Beep(frequency, duration);
//    }
//
//#else // Linux
//
//#include <cmath>
//#include <alsa/asoundlib.h>
//
//    void emitSound(int frequency, int duration) {
//        int rate = 44100; // Sampling rate
//        snd_pcm_t* handle;
//        snd_pcm_open(&handle, "default", SND_PCM_STREAM_PLAYBACK, 0);
//        snd_pcm_set_params(handle, SND_PCM_FORMAT_S16_LE, SND_PCM_ACCESS_RW_INTERLEAVED, 1, rate, 1, 500000);
//
//        short buf[rate * duration];
//
//        for (int i = 0; i < rate * duration; i++) {
//            int sample = 32760 * sin(2 * M_PI * frequency * i / rate);
//            buf[i] = sample;
//        }
//
//        snd_pcm_writei(handle, buf, rate * duration);
//        snd_pcm_close(handle);
//    }
//
//#endif 
	size_t free_bytes, total_bytes;
	hipError_t cuda_status = hipMemGetInfo(&free_bytes, &total_bytes);

	extern const unsigned long long TOtal_GPU_Bytes = (long long)free_bytes;

	// timing variables:
	  // fdmt time
	long long iFdmt_time = 0;
	// read && transform data time
	long long  iReadTransform_time = 0;
	// fft time
	long long  iFFT_time = 0;
	// detection time
	long long  iMeanDisp_time = 0;
	// detection time
	long long  iNormalize_time = 0;
	// total time
	long long  iTotal_time = 0;

	CChunk_gpu::~CChunk_gpu()
	{
		if (m_pd_arrcoh_dm)
		{
			hipFree(m_pd_arrcoh_dm);
		}
		if (m_pd_arr_dc)
		{
			hipFree(m_pd_arr_dc);
		}

		hipfftDestroy(m_fftPlanForward);
		hipfftDestroy(m_fftPlanInverse);
	}
	//-----------------------------------------------------------
	CChunk_gpu::CChunk_gpu() :CChunkB()
	{
		m_pd_arrcoh_dm = nullptr;
		m_pd_arr_dc = nullptr;
		
	}
	//-----------------------------------------------------------

	CChunk_gpu::CChunk_gpu(const  CChunk_gpu& R) :CChunkB(R)
	{
		hipMalloc(&m_pd_arrcoh_dm, R.m_coh_dm_Vector.size() * sizeof(double));
		hipMemcpy(m_pd_arrcoh_dm, R.m_pd_arrcoh_dm, m_coh_dm_Vector.size() * sizeof(double), hipMemcpyDeviceToDevice);

		hipMalloc(&m_pd_arr_dc, R.m_coh_dm_Vector.size() * R.m_nchan * R.m_nbin * sizeof(hipfftComplex));
		hipMemcpy(m_pd_arr_dc, R.m_pd_arr_dc,   R.m_coh_dm_Vector.size() * R.m_nchan * R.m_nbin * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
		m_Fdmt = R.m_Fdmt;

		hipfftDestroy(m_fftPlanForward);
		hipfftDestroy(m_fftPlanInverse);
		create_fft_plans();

	}
	//-------------------------------------------------------------------

	CChunk_gpu& CChunk_gpu::operator=(const CChunk_gpu& R)
	{
		if (this == &R)
		{
			return *this;
		}
		CChunkB:: operator= (R);

		hipMalloc(&m_pd_arrcoh_dm, R.m_coh_dm_Vector.size() * sizeof(double));
		hipMemcpy(m_pd_arrcoh_dm, R.m_pd_arrcoh_dm, m_coh_dm_Vector.size() * sizeof(double), hipMemcpyDeviceToDevice);

		hipMalloc(&m_pd_arr_dc, R.m_coh_dm_Vector.size() * R.m_nchan * R.m_nbin * sizeof(hipfftComplex));
		hipMemcpy(m_pd_arr_dc, R.m_pd_arr_dc, R.m_coh_dm_Vector.size() * R.m_nchan * R.m_nbin * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);

		m_Fdmt = R.m_Fdmt;
		hipfftDestroy(m_fftPlanForward);
		hipfftDestroy(m_fftPlanInverse);
		create_fft_plans();

		return *this;
	}
	//------------------------------------------------------------------
	CChunk_gpu::CChunk_gpu(
		const float Fmin
		, const float Fmax
		, const int npol
		, const int nchan		
		, const unsigned int len_sft
		, const int Block_id
		, const int Chunk_id
		, const  float d_max
		, const  float d_min
		, const int ncoherent
		, const float sigma_bound
		, const int length_sum_wnd
		, const int nbin
		, const int nfft
		, const int noverlap
		, const float tsamp) : CChunkB(Fmin
			, Fmax
			, npol
			, nchan			
			, len_sft
			, Block_id
			, Chunk_id
			, d_max
			, d_min
			, ncoherent
			, sigma_bound
			, length_sum_wnd
			, nbin
			, nfft
			, noverlap
			, tsamp)
	{
		// 1.
		const int ndm = m_coh_dm_Vector.size();
		// 1!

		hipMalloc(&m_pd_arrcoh_dm, ndm * sizeof(double));
		hipMemcpy(m_pd_arrcoh_dm, m_coh_dm_Vector.data(), ndm * sizeof(double), hipMemcpyHostToDevice);

		hipMalloc(&m_pd_arr_dc, ndm * m_nchan * m_nbin * sizeof(hipfftComplex));

		compute_chirp_channel( );
	}


void CChunk_gpu::compute_chirp_channel()
{
	// 1 preparations
	 double bw = m_Fmax - m_Fmin;
	int mbin = get_mbin();
	 double bw_sub = bw / m_nchan;
	 double bw_chan = bw_sub / m_len_sft;
	int ndm = m_coh_dm_Vector.size();
	//1!

	
	double* d_parr_freqs_chan = nullptr;
	hipMalloc(&d_parr_freqs_chan, m_nchan * m_len_sft * sizeof(double));

	const dim3 block_size(32, 1, 1);
	const dim3 gridSize((m_len_sft + block_size.x - 1) / block_size.x, m_nchan, 1);
	kernel_create_arr_freqs_chan << < gridSize, block_size >> >(d_parr_freqs_chan, m_len_sft, bw_chan, m_Fmin, bw_sub);
	// 3!

	//int lenarr1 = m_nchan * m_len_sft;// *sizeof(hipfftComplex));
	//std::vector<double> data1(lenarr1, 0);
	//hipMemcpy(data1.data(), d_parr_freqs_chan, lenarr1 * sizeof(double), hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
	//std::array<long unsigned, 1> leshape128{ lenarr1 };
	//npy::SaveArrayAsNumpy("d_parr_freqs_chan.npy", false, leshape128.size(), leshape128.data(), data1);
	

	double* d_parr_taper = nullptr;
	hipMalloc(&d_parr_taper, mbin * sizeof(double));
	double* d_parr_bin_freqs = nullptr;
	hipMalloc(&d_parr_bin_freqs, mbin * sizeof(double));

	const dim3 block_Size1(1024, 1, 1);
	const dim3 gridSize1((mbin + block_Size1.x - 1) / block_Size1.x, 1, 1);
	kernel_create_arr_bin_freqs_and_taper << < gridSize1, block_Size1 >> > (d_parr_bin_freqs, d_parr_taper, bw_chan, mbin);

	//int lenarr3 = mbin;// *sizeof(hipfftComplex));
	//std::vector<double> data3(lenarr3, 0);
	//hipMemcpy(data3.data(), d_parr_bin_freqs, lenarr3 * sizeof(double), hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
	//std::array<long unsigned, 1> leshape1{ lenarr3 };
	//npy::SaveArrayAsNumpy("parr_bin_freqs.npy", false, leshape1.size(), leshape1.data(), data3);

	//int lenarr4 = mbin;// *sizeof(hipfftComplex));
	//std::vector<double> data4(lenarr4, 0);
	//hipMemcpy(data4.data(), d_parr_taper, lenarr4 * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
	//std::array<long unsigned, 1> leshape2{ lenarr4 };
	//npy::SaveArrayAsNumpy("parr_taper.npy", false, leshape2.size(), leshape2.data(), data4);


	/*int deviceId;
	hipGetDevice(&deviceId);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceId);
	std::cout << "Maximum number of blocks per grid: " << deviceProp.maxGridSize[0] << std::endl;
	std::cout << "Maximum number of threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;*/
	auto start = std::chrono::high_resolution_clock::now();

	const dim3 block_Size2(512,2,1);
	const dim3 gridSize2((mbin + block_Size2.x - 1) / block_Size2.x, (m_len_sft * m_nchan + block_Size2.y - 1) / block_Size2.y, (ndm + block_Size2.z -1)/ block_Size2.z);
	kernel_create_arr_dc << < gridSize2, block_Size2 >> > (m_pd_arr_dc, m_pd_arrcoh_dm, d_parr_freqs_chan, d_parr_bin_freqs, d_parr_taper, ndm, m_nchan, m_len_sft, mbin);

	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "Time taken by function fncFdmtU_cu: " << duration.count() << " microseconds" << std::endl;
	hipFree(d_parr_freqs_chan);
	hipFree(d_parr_taper);
	hipFree(d_parr_bin_freqs);

	//int lenarr4 = ndm* m_nchan* m_len_sft* mbin;// *sizeof(hipfftComplex));
	//std::vector<complex<float>> data4(lenarr4, 0);
	//hipMemcpy(data4.data(), m_pd_arr_dc, lenarr4 * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
	//std::array<long unsigned, 1> leshape2{ lenarr4 };
	//npy::SaveArrayAsNumpy("arr_dc.npy", false, leshape2.size(), leshape2.data(), data4);


	// Generate FFT plans (batch in-place)
	create_fft_plans();


	

}
//--------------------------------------------------------------------------------------------
void CChunk_gpu::create_fft_plans()
{
	if (hipfftPlanMany(&m_fftPlanForward,1, &m_nbin,
		NULL, 1, m_nbin, // *inembed, istride, idist
		NULL, 1, m_nbin, // *onembed, ostride, odist
		HIPFFT_C2C, m_nfft * m_nchan * m_npol / 2) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	int mbin = get_mbin();
	
	checkCudaErrors(hipfftPlanMany(&m_fftPlanInverse, 1, &mbin, NULL,            1,      mbin, NULL,           1,         mbin, HIPFFT_C2C, m_len_sft * m_nfft * m_nchan * m_npol / 2));
}

//-------------------------------------------------------------------------------------------
__global__
void kernel_create_arr_dc  (hipfftComplex* parr_dc, double * parrcoh_dm,double* parr_freqs_chan, double* parr_bin_freqs, double *parr_taper
	, int ndm, int nchan, int len_sft, int mbin)
{
	//__shared__  double temp0[1];
	//__shared__  int i0[1];
	//printf("ququ");
	int ibin = blockIdx.x * blockDim.x + threadIdx.x;
	if (ibin >= mbin)
	{
		return;
	}
	//printf("ququ1");
	int num1 = blockIdx.y * blockDim.y + threadIdx.y;
	if (num1 >= nchan * len_sft)
	{
		return;
	}
	
	int ichan = num1 / len_sft;
	
	int isft = num1 % len_sft;

	int idm = blockIdx.z * blockDim.z + threadIdx.z;
	if (idm >= ndm)
	{
		return;
	}
	

	float temp0 = parr_freqs_chan[ichan * len_sft + isft];
	int i0 = idm * nchan * len_sft * mbin + ichan * len_sft * mbin + isft * mbin;


	double temp1 = parr_bin_freqs[ibin] / temp0;
	double phase_delay=(parrcoh_dm[idm] * temp1 * temp1 / (temp0 + parr_bin_freqs[ibin]) * 4.148808e9);
	double val_prD_int = 0;
	double t = -modf(phase_delay, &val_prD_int) * 2.0;
	double val_x = 0.0, val_y = 0.;
	
	
	sincospi(t, &val_y, &val_x);
	//printf("val_x = %f ;\n", (float)val_x);
	if (val_x > 0.0)
	{
		//printf("val_x = % ;\n", (float)val_x);
	}
	parr_dc[i0 + ibin].x = float(val_x * parr_taper[ibin]);
	parr_dc[i0 + ibin].y = float(val_y * parr_taper[ibin]);
	
}

//------------------------------------------------------------------------------------------
__global__
void kernel_create_arr_bin_freqs_and_taper(double* d_parr_bin_freqs, double* d_parr_taper,  double  bw_chan,  int mbin)
{	
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind >= mbin)
	{
		return;
	}
	double temp = -0.5 * bw_chan + (ind + 0.5) * bw_chan / mbin;
	d_parr_bin_freqs[ind] = temp;
	d_parr_taper[ind] = 1.0 / sqrt(1.0 + pow(temp / (0.47 * bw_chan), 80));
	
}
//----------------------------------------------------------------------------------------
__global__
void kernel_create_arr_freqs_chan(double* d_parr_freqs_chan, int len_sft, double bw_chan, double  Fmin, double bw_sub)
{
	int nchan = gridDim.y;
	int ichan = blockIdx.y;	
	int col_ind = blockIdx.x * blockDim.x + threadIdx.x;
	 double freqs_sub = Fmin + bw_sub * (0.5 + ichan);
	 double vi = (double)(col_ind % len_sft);
	 double temp = bw_chan * (vi - len_sft / 2.0 + 0.5);
	d_parr_freqs_chan[ichan * len_sft + col_ind] = freqs_sub + temp;
}
	////---------------------------------------------------
	bool CChunk_gpu::process(void* pcmparrRawSignalCur
		, std::vector<COutChunkHeader>* pvctSuccessHeaders, std::vector<std::vector<float>>* pvecImg)
	{
		// 1. 
		const int mbin = get_mbin();
		const int noverlap_per_channel = get_noverlap_per_channel();
		const int mbin_adjusted = get_mbin_adjusted();
		const int msamp = get_msamp();
		const int mchan = m_nchan * m_len_sft;
		// 1!
		
		// 2. Forward FFT execution
		checkCudaErrors(hipfftExecC2C(m_fftPlanForward, (hipfftComplex*)pcmparrRawSignalCur, (hipfftComplex*)pcmparrRawSignalCur, HIPFFT_FORWARD));

		
		//2!	

		//int lenarr4 = m_nfft * m_nchan * m_nbin * (m_npol / 2) / 2;// *sizeof(hipfftComplex));
		//std::vector<complex<float>> data4(lenarr4, 0);
		//hipMemcpy(data4.data(), (hipfftComplex*)pcmparrRawSignalCur, lenarr4 * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
		//hipDeviceSynchronize();
		//std::array<long unsigned, 1> leshape2{ lenarr4 };
		//npy::SaveArrayAsNumpy("pcmparrRawSignalFfted.npy", false, leshape2.size(), leshape2.data(), data4);
		//int ii = 0;

		//3. roll and normalize ffted signals
		dim3 treads_per_block(256, 1);
		dim3 blocks_per_grid((m_nbin + treads_per_block.x - 1) / treads_per_block.x, m_nfft * m_nchan * m_npol / 2);
		
		hipfftComplex* pcmparrRawSignalRolled = NULL;
		hipMalloc(&pcmparrRawSignalRolled, m_nfft * m_nchan * m_npol / 2 * m_nbin * sizeof(hipfftComplex));
		dim3 threads(1024, 1);
		dim3 blocks((m_nbin + threads.x - 1) / threads.x, m_nfft * m_nchan * m_npol / 2);
		roll_rows_and_normalize_kernel<<< blocks, threads>>>(pcmparrRawSignalRolled, (hipfftComplex*)pcmparrRawSignalCur, m_nfft * m_nchan * m_npol / 2, m_nbin, m_nbin / 2);

		//int lenarr4 = m_nfft * m_nchan * m_nbin * (m_npol / 2) / 2;// *sizeof(hipfftComplex));
		//std::vector<complex<float>> data4(lenarr4, 0);
		//hipfftComplex* pc = &pcmparrRawSignalRolled[lenarr4];
		//hipMemcpy(data4.data(), pcmparrRawSignalRolled, lenarr4 * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
		//hipDeviceSynchronize();
		//std::array<long unsigned, 1> leshape2{ lenarr4 };
		//npy::SaveArrayAsNumpy("pcmparrRawSignalFfted.npy", false, leshape2.size(), leshape2.data(), data4);
		//int ii = 0;
		
		//3!
		for (int idm = 0; idm < m_coh_dm_Vector.size(); ++idm)
		{
			dim3 threadsPerBlock(1024, 1, 1);
			dim3 blocksPerGrid((m_nchan * m_nbin + threadsPerBlock.x - 1) / threadsPerBlock.x, m_nfft, m_npol/2);
			element_wise_cufftComplex_mult_kernel<<< blocksPerGrid, threadsPerBlock>>>
				((hipfftComplex*)pcmparrRawSignalCur, pcmparrRawSignalRolled, &m_pd_arr_dc[m_nchan * m_nbin * idm], m_npol/2,m_nfft, m_nchan * m_nbin);


			 int threads1 = 1024;
			int blocks1 = (m_npol / 2 * m_nfft * m_nchan * m_nbin + threads1 - 1) / threads1;
			
			divide_cufftComplex_array_kernel <<<blocks1, threads1>>>((hipfftComplex*)pcmparrRawSignalCur, m_npol / 2* m_nfft * m_nchan * m_nbin,  ((float)m_nbin));
		//	int lenarr4 = m_nfft * m_nchan * m_nbin * (m_npol / 2) / 2;// *sizeof(hipfftComplex));
		//std::vector<complex<float>> data4(lenarr4, 0);
		//hipMemcpy(data4.data(), (hipfftComplex*)pcmparrRawSignalCur, lenarr4 * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
		//hipDeviceSynchronize();
		//std::array<long unsigned, 1> leshape2{ lenarr4 };
		//npy::SaveArrayAsNumpy("pcmparrRawSignalFfted.npy", false, leshape2.size(), leshape2.data(), data4);
		//int ii = 0;
			hipfftComplex* pcmparrRawSignalRolled1 = NULL;
			hipMalloc(&pcmparrRawSignalRolled1, m_nfft * m_nchan * m_npol / 2 * m_nbin * sizeof(hipfftComplex));
			int mbin = get_mbin();
			dim3 threads(256, 1);
			dim3 blocks((mbin + threads.x - 1) / threads.x, m_nfft * m_nchan * m_npol / 2 * m_len_sft);
			roll_rows_and_normalize_kernel << < blocks, threads >> > (pcmparrRawSignalRolled1, (hipfftComplex*)pcmparrRawSignalCur, m_nfft * m_nchan * m_npol / 2 *  m_len_sft, mbin, mbin);

			//int lenarr4 = m_nfft * m_nchan * m_nbin * (m_npol / 2) / 2;// *sizeof(hipfftComplex));
			//std::vector<complex<float>> data4(lenarr4, 0);
			//hipMemcpy(data4.data(), pcmparrRawSignalRolled1, lenarr4 * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
			//hipDeviceSynchronize();
			//std::array<long unsigned, 1> leshape2{ lenarr4 };
			//npy::SaveArrayAsNumpy("pcmparrRawSignalFfted.npy", false, leshape2.size(), leshape2.data(), data4);
			//int ii = 0;

			checkCudaErrors(hipfftExecC2C(m_fftPlanInverse, pcmparrRawSignalRolled1, pcmparrRawSignalRolled1, HIPFFT_BACKWARD));
			//checkCudaErrors(hipfftExecC2C(m_fftPlanInverse, (hipfftComplex*)pcmparrRawSignalCur, (hipfftComplex*)pcmparrRawSignalCur, HIPFFT_BACKWARD));
			
			
			blocks1 = (m_nfft * m_nchan * m_npol / 2 * m_nbin + threads1 - 1) / threads1;
			divide_cufftComplex_array_kernel << <blocks1, threads1 >> > (pcmparrRawSignalRolled1, m_nfft * m_nchan * m_npol / 2 * m_nbin, ((float)mbin));

			//int lenarr4 = m_nfft * m_nchan * m_nbin * (m_npol / 2) / 2;// *sizeof(hipfftComplex));
			//std::vector<complex<float>> data4(lenarr4, 0);
			//hipMemcpy(data4.data(), pcmparrRawSignalRolled1, lenarr4 * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
			//hipDeviceSynchronize();
			//std::array<long unsigned, 1> leshape2{ lenarr4 };
			//npy::SaveArrayAsNumpy("pcmparrRawSignalFfted.npy", false, leshape2.size(), leshape2.data(), data4);
			//int ii = 0;


			int noverlap_per_channel = get_noverlap_per_channel();
			int mbin_adjusted = get_mbin_adjusted();
			void* fbuf = NULL;
			hipMalloc(&fbuf, mbin_adjusted * m_nfft* m_nchan * m_len_sft * m_npol / 2 * sizeof(hipfftComplex));
			//void* fbuf = (hipfftComplex*)pcmparrRawSignalCur;
			
			dim3 threads_per_block1(1024, 1, 1);
			dim3 blocks_per_grid1(( mbin_adjusted + threads_per_block1.x - 1) / threads_per_block1.x, m_nchan * m_len_sft, m_nfft * m_npol/2);
			transpose_unpadd<<< blocks_per_grid1, threads_per_block1>>>
				((hipfftComplex*)fbuf, pcmparrRawSignalRolled1, m_nfft, noverlap_per_channel
				, mbin_adjusted, m_nchan, m_len_sft, mbin);

			int lenarr4 = m_nfft * mbin_adjusted * m_nchan * m_len_sft * m_npol / 2 / 2;// *sizeof(hipfftComplex));
			std::vector<complex<float>> data4(lenarr4, 0);
			hipMemcpy(data4.data(), fbuf, lenarr4 * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			std::array<long unsigned, 1> leshape2{ lenarr4 };
			npy::SaveArrayAsNumpy("pcmparrRawSignalFfted.npy", false, leshape2.size(), leshape2.data(), data4);
			int ii = 0;

			hipFree(pcmparrRawSignalRolled1);
			hipFree(fbuf);
		}
		hipFree(pcmparrRawSignalRolled);
		return true;
	}
//-----------------------------------------------------------------------------------
//		
__global__	void  transpose_unpadd(hipfftComplex* fbuf, hipfftComplex* arin,int nfft,  int noverlap_per_channel
	, int mbin_adjusted, const int nchan, const int nlen_sft, int mbin)	
{
	int  ibin = blockIdx.x * blockDim.x + threadIdx.x;
	if (!(ibin < mbin_adjusted))
	{
		return;
	}
	int ipol = blockIdx.z / nfft;
	int ifft = blockIdx.z % nfft;
	int ichan = blockIdx.y / nlen_sft;
	int ilen_sft = blockIdx.y % nlen_sft;
	int ibin_adjusted = ibin + noverlap_per_channel;
	int isamp = ibin + mbin_adjusted * ifft;
	 
	
	// Select bins from valid region and reverse the frequency axis		
	// float temp = arin[ifft * nchan * nlen_sft * mbin + (nchan - ichan - 1) * nlen_sft * mbin + ilen_sft * mbin + ibin_adjusted].y;
	 //printf("%f \n", temp);
	
	fbuf[ipol * mbin_adjusted * nchan * nlen_sft  +  isamp * nchan * nlen_sft + ichan * nlen_sft + nlen_sft - ilen_sft - 1].x =
		arin[ipol * nfft * nchan * nlen_sft * mbin + ifft * nchan * nlen_sft * mbin + (nchan - ichan - 1) * nlen_sft * mbin + ilen_sft * mbin + ibin_adjusted].x;
	fbuf[ipol * mbin_adjusted * nchan * nlen_sft +  isamp * nchan * nlen_sft + ichan * nlen_sft + nlen_sft - ilen_sft - 1].y =
		arin[ipol *  nfft * nchan * nlen_sft * mbin +  ifft * nchan * nlen_sft * mbin + (nchan - ichan - 1) * nlen_sft * mbin + ilen_sft * mbin + ibin_adjusted].y;
	//for (int ifft = 0; ifft < nfft; ++ifft)
	//{
	//	for (int ichan = 0; ichan < nchan; ++ichan)
	//	{
	//		for (int ilen_sft = 0; ilen_sft < nlen_sft; ++ilen_sft)
	//		{
	//			for (int ibin = 0; ibin < mbin_adjusted; ++ibin)
	//			{
	//				int ibin_adjusted = ibin + noverlap_per_channel;
	//				int isamp = ibin + mbin_adjusted * ifft;


	//				// Select bins from valid region and reverse the frequency axis					
	//				fbuf[isamp * nchan * nlen_sft + ichan * nlen_sft + nlen_sft - ilen_sft - 1][0] =
	//					arin[ifft * nchan * nlen_sft * mbin + (nchan - ichan - 1) * nlen_sft * mbin + ilen_sft * mbin + ibin_adjusted][0];
	//				fbuf[isamp * nchan * nlen_sft + ichan * nlen_sft + nlen_sft - ilen_sft - 1][1] =
	//					arin[ifft * nchan * nlen_sft * mbin + (nchan - ichan - 1) * nlen_sft * mbin + ilen_sft * mbin + ibin_adjusted][1];


	//			}
	//		}
	//	}
	//}
}
	
//
//	void  CChunk_cpu::transpose_unpadd(fftwf_complex* arin, fftwf_complex* fbuf)
//	{
//		int noverlap_per_channel = get_noverlap_per_channel();
//		int mbin_adjusted = get_mbin_adjusted();
//		const int nsub = m_nchan;
//		const int nchan = m_len_sft;
//		const int mbin = get_mbin();
//#pragma omp parallel
//		{
//			for (int ifft = 0; ifft < m_nfft; ++ifft)
//			{
//				for (int ichan = 0; ichan < nchan; ++ichan)
//				{
//					for (int ibin = 0; ibin < mbin_adjusted; ++ibin)
//					{
//						int ibin_adjusted = ibin + noverlap_per_channel;
//						int isamp = ibin + mbin_adjusted * ifft;
//						int num = 0;
//						for (int isub = 0; isub < nsub; ++isub)
//						{
//							// Select bins from valid region and reverse the frequency axis					
//							fbuf[isamp * nsub * nchan + isub * nchan + nchan - ichan - 1][0] =
//								arin[ifft * nsub * nchan * mbin + (nsub - isub - 1) * nchan * mbin + ichan * mbin + ibin_adjusted][0];
//							fbuf[isamp * nsub * nchan + isub * nchan + nchan - ichan - 1][1] =
//								arin[ifft * nsub * nchan * mbin + (nsub - isub - 1) * nchan * mbin + ichan * mbin + ibin_adjusted][1];
//							++num;
//						}
//					}
//				}
//			}
//		}
//	}
	//-----------------------------------------------------------------------
	__global__ void  divide_cufftComplex_array_kernel(hipfftComplex* d_arr, int len, float val)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= len)
		{
			return;
		}
		d_arr[idx].x /= val;
		d_arr[idx].y /= val;
	}
//-------------------------------------------------------------------------------------------------
	//---------------------------------------------------------------
	__global__
		void scaling_kernel(hipfftComplex* data, long long element_count, float scale)
	{
		const int tid = threadIdx.x;
		const int stride = blockDim.x;
		for (long long i = tid; i < element_count; i += stride)
		{
			data[i].x *= scale;
			data[i].y *= scale;
		}
	}
__global__ void  element_wise_cufftComplex_mult_kernel(hipfftComplex * d_arrOut, hipfftComplex * d_arrInp0, hipfftComplex * d_arrInp1
	, int npol, int nfft, int dim2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dim2)
	{
		return;
	}
	//printf("ff\n");
	int ipol = blockIdx.z;
	int ifft = blockIdx.y;
	int ibegin = ipol * nfft * dim2 + ifft * dim2;
	d_arrOut[ibegin + idx].x = d_arrInp0[ibegin + idx].x* d_arrInp1[idx].x - d_arrInp0[ibegin + idx].y * d_arrInp1[idx].y;
	d_arrOut[ibegin + idx].y = d_arrInp0[ibegin + idx].x* d_arrInp1[idx].y + d_arrInp0[ibegin + idx].y * d_arrInp1[idx].x;
	//printf("%f  %f\n", d_arrOut[ibegin + idx].x, d_arrOut[ibegin + idx].y);
	
}
	
	//--------------------------------------------------------------------------------

	__global__ void roll_rows_and_normalize_kernel(hipfftComplex* arr_rez, hipfftComplex* arr, int rows, int cols, int shift)
	{

		int idx0 = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx0 >= cols)
		{
			return;
		}
		int ind_new = blockIdx.y * cols + (idx0 + shift) % cols;
		int ind = blockIdx.y * cols + idx0;
		arr_rez[ind_new].x = arr[ind].x /*/cols*/;
		arr_rez[ind_new].y = arr[ind].y/*/cols*/ ;

	}

//-----------------------------------------------------------------

long long CChunk_gpu::calcLenChunk_(CTelescopeHeader header, const int nsft
	, const float pulse_length, const float d_max)
{
	const int nchan_actual = nsft * header.m_nchan;

	long long len = 0;
	for (len = 1 << 9; len < 1 << 30; len <<= 1)
	{
		// iMaxDt !!!!! ???????????????? 
		CFdmtGpu fdmt(
			header.m_centfreq - header.m_chanBW * header.m_nchan / 2.
			, header.m_centfreq + header.m_chanBW * header.m_nchan / 2.
			, nchan_actual
			, len
			, nchan_actual
		);

		
		long long size0 = fdmt.calcSizeAuxBuff_fdmt_();
		long long size_fdmt_inp = fdmt.calc_size_input();
		long long size_fdmt_out = fdmt.calc_size_output();
		long long size_fdmt_norm = size_fdmt_out;
		long long irest = header.m_nchan * header.m_npol * header.m_nbits / 8 // input buff
			+ header.m_nchan * header.m_npol / 2 * sizeof(hipfftComplex)
			+ 3 * header.m_nchan * header.m_npol * sizeof(hipfftComplex) / 2
			+ 2 * header.m_nchan * sizeof(float);
		irest *= len;

		long long rez = size0 + size_fdmt_inp + size_fdmt_out + size_fdmt_norm + irest;
		if (rez > 0.98 * TOtal_GPU_Bytes)
		{
			return len / 2;
		}

	}
	return -1;
}

//---------------------------------------------------
//bool CChunk_gpu::fnCChunk_gpuProcessing_gpu(hipfftComplex* pcmparrRawSignalCur
//	, void* pAuxBuff_fdmt
//	, hipfftComplex* pcarrTemp
//	, hipfftComplex* pcarrCD_Out
//	, hipfftComplex* pcarrBuff
//	, float* pInpOutBuffFdmt, fdmt_type_* d_arrfdmt_norm
//	, const int IDeltaT, hipfftHandle plan0, hipfftHandle plan1
//	, structOutDetection* pstructOut
//	, float* pcoherentDedisp)
//{
//	// 1. installation of pointers	for pAuxBuff_the_rest
//	fdmt_type_* d_parr_fdmt_inp = (fdmt_type_*)pInpOutBuffFdmt; //4	
//	fdmt_type_* d_parr_fdmt_out = (fdmt_type_*)pInpOutBuffFdmt + m_nbin * m_nchan;
//	//// !1
//
//	 /*std::vector<std::complex<float>> data2(m_nbin, 0);
//	hipMemcpy(data2.data(), pcmparrRawSignalCur, m_nbin * sizeof(std::complex<float>),
//		hipMemcpyDeviceToHost);
//	hipDeviceSynchronize();*/
//	//std::array<long unsigned, 1> leshape127{ LEnChunk };
//	//npy::SaveArrayAsNumpy("ffted.npy", false, leshape127.size(), leshape127.data(), data);
//	auto start = std::chrono::high_resolution_clock::now();
//
//
//	// 2. create FFT	
//	//hipfftComplex* pcmparr_ffted = NULL;
//	//checkCudaErrors(hipMallocManaged((void**)&pcmparr_ffted, m_nbin * m_nchan * m_npol/2 * sizeof(hipfftComplex)));
//	//checkCudaErrors(hipfftExecC2C(plan0, pcmparrRawSignalCur, pcmparr_ffted, HIPFFT_FORWARD));
//	checkCudaErrors(hipfftExecC2C(plan0, pcmparrRawSignalCur, pcmparrRawSignalCur, HIPFFT_FORWARD));
//
//	// !2
//
//	auto end = std::chrono::high_resolution_clock::now();
//	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
//	iFFT_time += duration.count();
//
//	/*std::vector<std::complex<float>> data(m_nbin * m_nchan * m_npol/2, 0);
//	hipMemcpy(data.data(), pcmparrRawSignalCur, m_nbin * m_nchan * m_npol/2 * sizeof(std::complex<float>),
//		hipMemcpyDeviceToHost);
//	hipDeviceSynchronize();*/
//
//
//	// 3.
//	/*float chanBW = (m_Fmax - m_Fmin) / m_nchan;
//	float f1 = chanBW + m_Fmin;
//	float valConversionConst = DISPERSION_CONSTANT * (1. / (m_Fmin * m_Fmin) - 1. / (f1 * f1)) * chanBW;*/
//	float valConversionConst = DISPERSION_CONSTANT * (1. / (m_Fmin * m_Fmin) - 1. / (m_Fmax * m_Fmax)) * (m_Fmax - m_Fmin);
//	float valN_d = m_d_max * valConversionConst;
//	const int N_p = m_len_sft * m_nchan;
//	int n_coherent = int(ceil(valN_d / (N_p * N_p)));
//	cout << " n_coherent = " << n_coherent << endl;
//	// !3
//
//
//	structOutDetection* pstructOutCur = NULL;
//	checkCudaErrors(hipMallocManaged((void**)&pstructOutCur, sizeof(structOutDetection)));
//	hipDeviceSynchronize();
//	pstructOutCur->snr = 1. - FLT_MAX;
//	pstructOutCur->icol = -1;
//	pstructOut->snr = m_sigma_bound;
//	//// 4. main loop
//	const int IMaxDT = N_p;
//
//	float coherent_d = -1.;
//	bool breturn = false;
//	for (int iouter_d = 0; iouter_d < n_coherent; ++iouter_d)
//		//for (int iouter_d = 31; iouter_d < 32; ++iouter_d)
//	{
//
//		 float valcur_coherent_d = ((float)iouter_d) * ((float)m_d_max / ((float)n_coherent));
//		cout << "Chunk=  " << m_Chunk_id << "; chunk= " << iouter_d << "; iter= " << iouter_d << "; coherent_d = " << valcur_coherent_d << endl;
//
//		/*std::vector<std::complex<float>> data(m_nbin * m_nchan * m_npol /2, 0);
//		hipMemcpy(data.data(), pcmparrRawSignalCur, m_nbin * m_nchan * m_npol / 2 * sizeof(std::complex<float>),
//		hipMemcpyDeviceToHost);
//		hipDeviceSynchronize();*/
//
//		// fdmt input matrix computation
//		calcFDMT_Out_gpu(d_parr_fdmt_out, pcmparrRawSignalCur, pcarrCD_Out
//			, pcarrTemp, d_parr_fdmt_inp
//			, IMaxDT, DISPERSION_CONSTANT * valcur_coherent_d
//			, pAuxBuff_fdmt, IDeltaT, plan0, plan1, pcarrBuff);
//		// !
//		///*float* parr_fdmt_out = (float*)malloc(m_nbin * m_nchan* sizeof(float));
//		//hipMemcpy(parr_fdmt_out, d_parr_fdmt_out, m_nbin * m_nchan * sizeof(float), hipMemcpyDeviceToHost);
//		//float valmax = -0., valmin = 0.;
//		//unsigned int iargmax = -1, iargmin = -1;
//		//findMaxMinOfArray(parr_fdmt_out, m_nbin * m_nchan, &valmax, &valmin
//		//	, &iargmax, &iargmin);
//		//float* arrfdmt_norm = (float*)malloc(m_nbin * m_nchan * sizeof(float));
//		//hipMemcpy(arrfdmt_norm, d_arrfdmt_norm, m_nbin * m_nchan * sizeof(float), hipMemcpyDeviceToHost);
//		//float valmax1 = -0., valmin1 = 0.;
//		//unsigned int iargmax1 = -1, iargmin1 = -1;
//		//findMaxMinOfArray(arrfdmt_norm, m_nbin * m_nchan, &valmax1, &valmin1
//		//	, &iargmax1, &iargmin1);
//		//free(parr_fdmt_out);
//		//free(arrfdmt_norm);*/
//
//		//
//		const int Rows = m_len_sft * m_nchan;
//		const int Cols = m_nbin / m_len_sft;
//		const dim3 Totalsize(1024, 1, 1);
//		const dim3 gridSize((Cols + Totalsize.x - 1) / Totalsize.x, Rows, 1);
//		float* d_pAuxArray = (float*)d_parr_fdmt_inp;
//		int* d_pAuxNumArray = (int*)(d_pAuxArray + gridSize.x * gridSize.y);
//		int* d_pWidthArray = d_pAuxNumArray + gridSize.x * gridSize.y;
//		detect_signal_gpu(d_parr_fdmt_out, d_arrfdmt_norm, Rows
//			, Cols, m_length_sum_wnd, gridSize, Totalsize
//			, d_pAuxArray, d_pAuxNumArray, d_pWidthArray, pstructOutCur);
//		if ((*pstructOutCur).snr >= (*pstructOut).snr)
//		{
//			(*pstructOut).snr = (*pstructOutCur).snr;
//			(*pstructOut).icol = (*pstructOutCur).icol;
//			(*pstructOut).irow = (*pstructOutCur).irow;
//			(*pstructOut).iwidth = (*pstructOutCur).iwidth;
//
//			*pcoherentDedisp = valcur_coherent_d;
//
//			std::cout << "SNR = " << (*pstructOut).snr << endl;
//			std::cout << "ROW ARGMAX = " << (*pstructOut).irow << endl;
//			std::cout << "COLUMN ARGMAX = " << (*pstructOut).icol << endl;
//
//			int frequency = 1500; // Frequency in hertz
//			int duration = 500;   // Duration in milliseconds
//			emitSound(frequency, duration / 4);
//			emitSound(frequency + 500, duration / 2);
//			d_pAuxArray = NULL;
//			d_pAuxNumArray = NULL;
//
//			d_pWidthArray = NULL;
//
//			breturn = true;
//		}
//		//
//		///*std::vector<float> data(LEnChunk, 0);
//		//hipMemcpy(data.data(), parr_fdmt_out, LEnChunk * sizeof(float),
//		//	hipMemcpyDeviceToHost);
//		//hipDeviceSynchronize();*/
//	}
//	hipFree(pstructOutCur);
//	return breturn;
//}
//--------------------------------------------------------------

//--------------------------------------------------------------------
//INPUT:
//1. pcarrTemp - complex array with total length  = m_nbin * (m_npol/2)* m_nchan
// pcarrTemp can be interpreted as matrix, consisting of  m_nchan *(m_npol/2) rows
// each row consists of m_len_sft subrows corresponding to m_len_sft subfrequencies
// 2.pAuxBuff - auxillary buffer to compute mean and dispersions of each row ofoutput matrix d_parr_fdmt_inp
//OUTPUT:
//d_parr_fdmt_inp - matrix with dimensions (m_nchan*m_len_sft) x (m_nbin/m_len_sft)
// d_parr_fdmt_inp[i][j] = 
//
void CChunk_gpu::calc_fdmt_inp(fdmt_type_* d_parr_fdmt_inp, hipfftComplex* pcarrTemp
	, float*pAuxBuff)
{	
	
	/*dim3 threadsPerChunk(TILE_DIM, TILE_DIM, 1);
	dim3 ChunksPerGrid((m_len_sft + TILE_DIM - 1) / TILE_DIM, (m_nbin / m_len_sft + TILE_DIM - 1) / TILE_DIM, m_nchan);
	size_t sz = TILE_DIM * (TILE_DIM + 1) * sizeof(float);
	float* d_parr_fdmt_inp_flt = pAuxBuff;	
	calcPowerMtrx_kernel << < ChunksPerGrid, threadsPerChunk, sz >> > (d_parr_fdmt_inp_flt, m_nbin/ m_len_sft, m_len_sft, m_npol, pcarrTemp);
	hipDeviceSynchronize();*/


	//std::vector<float> data0(m_nbin, 0);
	//hipMemcpy(data0.data(), d_parr_fdmt_inp_flt, m_nbin * sizeof(float),
	//	hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
	//float valmax = 0., valmin = 0.;
	//unsigned int iargmax = 0, iargmin = 0;
	//findMaxMinOfArray(data0.data(), data0.size(), &valmax, &valmin
	//	, &iargmax, &iargmin);
	float* d_parr_fdmt_temp = pAuxBuff;
	dim3 threadsPerChunk(1024, 1, 1);
	dim3 ChunksPerGrid((m_nbin * threadsPerChunk.x - 1) / threadsPerChunk.x, m_nchan, 1);
	calcPartSum_kernel<<< ChunksPerGrid, threadsPerChunk>>>(d_parr_fdmt_temp, m_nbin, m_npol/2, pcarrTemp);
	hipDeviceSynchronize();

	/*std::vector<float> data0(m_nbin, 0);
	hipMemcpy(data0.data(), d_parr_fdmt_temp, m_nbin * sizeof(float),
		hipMemcpyDeviceToHost);*/

	float* d_parr_fdmt_inp_flt = d_parr_fdmt_temp + m_nbin * m_nchan;
	dim3 threadsPerBlock1(TILE_DIM, TILE_DIM, 1);
	dim3 blocksPerGrid1((m_len_sft + TILE_DIM - 1) / TILE_DIM, (m_nbin / m_len_sft + TILE_DIM - 1) / TILE_DIM, m_nchan);
	size_t sz = TILE_DIM * (TILE_DIM + 1) * sizeof(float);
	multiTransp_kernel << < blocksPerGrid1, threadsPerBlock1, sz >> > (d_parr_fdmt_inp_flt, m_nbin / m_len_sft, m_len_sft, d_parr_fdmt_temp);
	hipDeviceSynchronize();

	/*std::vector<float> data6(m_nbin, 0);
	hipMemcpy(data6.data(), d_parr_fdmt_inp_flt, m_nbin * sizeof(float),
		hipMemcpyDeviceToHost);*/
	int nFdmtRows = m_nchan * m_len_sft;
	int nFdmtCols = m_nbin / m_len_sft;
	float* d_arrRowMean = (float*)pcarrTemp;
	float* d_arrRowDisp = d_arrRowMean + nFdmtRows;
	
	
	auto start = std::chrono::high_resolution_clock::now();

	// Calculate mean and variance
	float* pval_mean = d_arrRowDisp + nFdmtRows;
	float* pval_stdDev = pval_mean + 1;
	float* pval_dispMean = pval_stdDev + 1;
	float* pval_dispStd = pval_dispMean + 1;
	

	ChunksPerGrid = nFdmtRows;
	int treadsPerChunk = calcThreadsForMean_and_Disp(nFdmtCols);
	size_t sz1 = (2 * sizeof(float) + sizeof(int)) * treadsPerChunk;
	// 1. calculations mean values and dispersions for each row of matrix d_parr_fdmt_inp_flt
	// d_arrRowMean - array contents  mean values of each row of input matrix pcarrTemp
	// d_arrRowDisp - array contents  dispersions of each row of input matrix pcarrTemp
	
	calcRowMeanAndDisp << < ChunksPerGrid, treadsPerChunk, sz1 >> > (d_parr_fdmt_inp_flt, nFdmtRows, nFdmtCols, d_arrRowMean, d_arrRowDisp);
	hipDeviceSynchronize();

	/*std::vector<float> data4(nRows, 0);
	hipMemcpy(data4.data(), d_arrRowDisp, nRows * sizeof(float),
		hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	std::vector<float> data5(nRows, 0);
	hipMemcpy(data5.data(), d_arrRowMean, nRows * sizeof(float),
		hipMemcpyDeviceToHost);
	hipDeviceSynchronize();*/

	//float* parr_fdmt_inp_flt = (float*)malloc(nRows* nCols *sizeof(float));
	//hipMemcpy(parr_fdmt_inp_flt, d_parr_fdmt_inp_flt, nRows * nCols * sizeof(float), hipMemcpyDeviceToHost);
	//float* arrM = (float*)malloc(nRows * sizeof(float));
	//float* arrD = (float*)malloc(nRows * sizeof(float));
	//memset(arrM, 0, nRows * sizeof(float));
	//memset(arrD, 0, nRows * sizeof(float));
	//for (int i = 0; i < nRows; ++i)
	//{
	//	for (int j = 0; j < nCols; ++j)
	//	{
	//		arrM[i] += parr_fdmt_inp_flt[i * nCols + j];
	//		arrD[i] += parr_fdmt_inp_flt[i * nCols + j] * parr_fdmt_inp_flt[i * nCols + j];
	//	}
	//	arrM[i] = arrM[i] / ((float)nCols);
	//	arrD[i] = arrD[i] / ((float)nCols) - arrM[i] * arrM[i];

	//}


	//free(parr_fdmt_inp_flt);
	//free(arrM);
	//free(arrD);
	// 2. calculations mean value and standart deviation for full matrix pcarrTemp
	// it is demanded to normalize matrix pcarrTemp
	ChunksPerGrid = 1;
	treadsPerChunk = calcThreadsForMean_and_Disp(nFdmtRows);
	sz = treadsPerChunk * (2 * sizeof(float) + sizeof(int));
	kernel_OneSM_Mean_and_Std << <ChunksPerGrid, treadsPerChunk, sz >> > (d_arrRowMean, d_arrRowDisp, nFdmtRows
		, pval_mean, pval_stdDev);
	hipDeviceSynchronize();

	//

	float mean = -1., disp = -1.;
	hipMemcpy(&mean, pval_mean, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&disp, pval_stdDev, sizeof(float), hipMemcpyDeviceToHost);

	//// check up
	//float* arrmean = (float*)malloc(nRows * sizeof(float));
	//float* arrdisp = (float*)malloc(nRows * sizeof(float));
	//hipMemcpy(arrmean, d_arrRowMean, nRows * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(arrdisp, d_arrRowDisp, nRows * sizeof(float), hipMemcpyDeviceToHost);
	//float sum = 0.;
	//for (int i = 0; i < nRows; ++i)
	//{
	//	sum += arrmean[i];
	//}
	//sum = sum / ((float)nRows);

	//float disp1 = 0;
	//for (int i = 0; i < nRows; ++i)
	//{
	//	disp1 += arrmean[i] * arrmean[i] + arrdisp[i];// (arrmean[i] - sum)* (arrmean[i] - sum);
	//}
	//disp1 = disp1/ ((float)nRows) - sum*sum;

	//free(arrmean);
	//free(arrdisp);


	// 3. calculations mean value and standart deviation for array d_arrRowDisp
	// it is demanded to clean out tresh from matrix pcarrTemp
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	iMeanDisp_time += duration.count();
	
	
	
	int threads = 128;
	calculateMeanAndSTD_for_oneDimArray_kernel << <1, threads, threads * 2 * sizeof(float) >> > (d_arrRowDisp, nFdmtRows, pval_dispMean, pval_dispStd);
	hipDeviceSynchronize();

	/*float hval_dispMean = -1;
	float hval_dispStd = -1;
	hipMemcpy(&hval_dispMean, pval_dispMean, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&hval_dispStd, pval_dispStd, sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	pval_dispMean = NULL;
	pval_dispStd = NULL;*/

	// 4.Clean and normalize array
	const dim3 Totalsize(256, 1, 1);
	const dim3 gridSize(1, nFdmtRows, 1);
	
	

	normalize_and_clean << < gridSize, Totalsize >> >
		(d_parr_fdmt_inp, d_parr_fdmt_inp_flt, nFdmtRows, nFdmtCols
		, pval_mean, pval_stdDev, d_arrRowDisp, pval_dispMean, pval_dispStd  );	
	hipDeviceSynchronize();

	float* parr_fdmt_inp = (float*)malloc(nFdmtRows * nFdmtCols * sizeof(float));
	hipMemcpy(parr_fdmt_inp, d_parr_fdmt_inp, nFdmtRows* nFdmtCols * sizeof(float), hipMemcpyDeviceToHost);

	//float valmax = -0., valmin = 0.;
	//unsigned int iargmax = -1, iargmin = -1;
	//findMaxMinOfArray(parr_fdmt_inp, nRows * nCols, &valmax,  &valmin
	//	, &iargmax, &iargmin);

	//auto end1 = std::chrono::high_resolution_clock::now();
	//auto duration1 = std::chrono::duration_cast<std::chrono::microseconds>(end1 - end);
	//iNormalize_time += duration1.count();
	//free(parr_fdmt_inp);

	d_arrRowMean = NULL;
	d_arrRowDisp = NULL;
	pval_mean = NULL;	
	pval_stdDev = NULL;
}
//--------------------------------------
void CChunk_gpu::set_chunkid(const int nC)
{
	m_Chunk_id = nC;
}
//--------------------------------------
void CChunk_gpu::set_blockid(const int nC)
{
	m_Block_id = nC;
}
//-------------------------------------------------------------------
__device__
float fnc_norm2(hipfftComplex* pc)
{
	return ((*pc).x * (*pc).x + (*pc).y * (*pc).y);
}




//----------------------------------------------------

__global__
void calcMultiTransposition_kernel(fdmt_type_* output, const int height, const int width, fdmt_type_* input)
{
	__shared__ fdmt_type_ tile[TILE_DIM][TILE_DIM + 1]; // Shared memory tile
	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int ichan = blockIdx.z;
	// Transpose data from global to shared memory
	if (x < width && y < height)
	{
		tile[threadIdx.y][threadIdx.x] = input[ichan * height * width + y * width + x];
	}
	__syncthreads();

	// Calculate new indices for writing to output
	x = blockIdx.y * TILE_DIM + threadIdx.x;
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	// Transpose data from shared to global memory
	if (x < height && y < width)
	{
		output[ichan * height * width + y * height + x] = tile[threadIdx.x][threadIdx.y];
	}
}
//------------------------------------------
__global__
void calcPartSum_kernel(float* d_parr_out, const int lenChunk, const int npol_physical, hipfftComplex* d_parr_inp)
{
	int ichan = blockIdx.y;
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind < lenChunk)
	{
		float sum = 0;
		for (int i = 0; i < npol_physical; ++i)
		{
			sum += fnc_norm2(&d_parr_inp[(ichan * npol_physical + i) * lenChunk + ind]);
		}
		d_parr_out[ichan * lenChunk + ind] = sum;
	}
}
//------------------------------------------
__global__
void calcPowerMtrx_kernel(float* output, const int height, const int width, const int npol, hipfftComplex* input)
{
	__shared__ float tile[TILE_DIM][TILE_DIM + 1]; // Shared memory tile
	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int ichan = blockIdx.z;
	// Transpose data from global to shared memory
	if (x < width && y < height)
	{
		float sum = 0.;
		for (int i = 0; i < npol / 2; ++i)
		{
			sum += fnc_norm2(&input[(ichan * npol / 2 + i) * height * width + y * width + x]);
		}

		tile[threadIdx.y][threadIdx.x] = sum;
	}
	__syncthreads();

	// Calculate new indices for writing to output
	x = blockIdx.y * TILE_DIM + threadIdx.x;
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	// Transpose data from shared to global memory
	if (x < height && y < width) {
		output[ichan * height * width + y * height + x] = tile[threadIdx.x][threadIdx.y];
	}
}
//------------------------------------------
__global__
void multiTransp_kernel(float* output, const int height, const int width, float* input)
{
	__shared__ float tile[TILE_DIM][TILE_DIM + 1]; // Shared memory tile

	int numchan = blockIdx.z;
	float* pntInp = &input[numchan * height * width];
	float* pntOut = &output[numchan * height * width];

	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;

	// Transpose data from global to shared memory
	if (x < width && y < height) {
		tile[threadIdx.y][threadIdx.x] = pntInp[y * width + x];
	}

	__syncthreads();

	// Calculate new indices for writing to output
	x = blockIdx.y * TILE_DIM + threadIdx.x;
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	// Transpose data from shared to global memory
	if (x < height && y < width) {
		pntOut[y * height + x] = tile[threadIdx.x][threadIdx.y];
	}
}

//---------------------------------------------------------------

__global__ 
void normalize_and_clean(fdmt_type_* parrOut, float* d_arr, const int NRows, const int NCols
	,float *pmean, float *pstd, float* d_arrRowDisp, float *pmeanDisp, float *pstdDisp)
{
	__shared__ int sbad[1];
	unsigned int i = threadIdx.x;
	unsigned int irow = blockIdx.y;
	if (i >= NCols)
	{
		return;
	}
	if (fabs(d_arrRowDisp[irow] - *pmeanDisp) > 4. * (*pstdDisp))
	{
		sbad[0] = 1;
	}
	else
	{
		sbad[0] = 0;
	}
	//--------------------------------
	if (sbad[0] == 1)
	{
		while (i < NCols)
		{
			parrOut[irow * NCols + i] = 0;
			i += blockDim.x;
		}
	}
	else
	{
		while (i < NCols)
		{
			parrOut[irow * NCols + i] = (fdmt_type_)((d_arr[irow * NCols + i] - (*pmean) )/((*pstd )));
			i += blockDim.x;
		}
	}
	

}

//-------------------------------------------
void CChunk_gpu::preparations_and_memoryAllocations(CTelescopeHeader header
	, const float pulse_length
	, const float d_max
	, const float sigma_bound
	, const int length_sum_wnd
	, int* pLenChunk
	, hipfftHandle* pplan0, hipfftHandle* pplan1, CFdmtU* pfdmt, char** d_pparrInput, hipfftComplex** ppcmparrRawSignalCur
	, void** ppAuxBuff_fdmt, fdmt_type_** d_parrfdmt_norm
	, hipfftComplex** ppcarrTemp
	, hipfftComplex** ppcarrCD_Out
	, hipfftComplex** ppcarrBuff, char** ppInpOutBuffFdmt,  CChunk_gpu** ppChunk)
{
	//hipError_t cudaStatus;
	//const float VAlFmin = header.m_centfreq - ((float)header.m_nchan) * header.m_chanBW / 2.0;
	//const float VAlFmax = header.m_centfreq + ((float)header.m_nchan) * header.m_chanBW / 2.0;
	//// 3.2 calculate standard len_sft and LenChunk    
	//const int len_sft = calc_len_sft(fabs(header.m_chanBW), pulse_length);
	//*pLenChunk = calcLenChunk_(header, len_sft, pulse_length, d_max);


	//// 3.3 cuFFT plans preparations

	//hipfftCreate(pplan0);
	//checkCudaErrors(hipfftPlan1d(pplan0, *pLenChunk, HIPFFT_C2C, header.m_nchan * header.m_npol / 2));


	//
	//hipfftCreate(pplan1);
	//checkCudaErrors(hipfftPlan1d(pplan1, len_sft, HIPFFT_C2C, (*pLenChunk) * header.m_nchan * header.m_npol / 2 / len_sft));
	//


	//// !3

	//// 4. memory allocation in GPU
	//// total number of downloding bytes to each file:
	//const long long QUantDownloadingBytesForChunk = (*pLenChunk) * header.m_nchan / 8 * header.m_nbits* header.m_npol;

	//const long long QUantBlockComplexNumbers = (*pLenChunk) * header.m_nchan * header.m_npol / 2;



	//checkCudaErrors(hipMallocManaged((void**)d_pparrInput, QUantDownloadingBytesForChunk * sizeof(char)));


	//checkCudaErrors(hipMalloc((void**)ppcmparrRawSignalCur, QUantBlockComplexNumbers * sizeof(hipfftComplex)));
	//// 2!

	//

	//// 4.memory allocation for auxillary buffer for fdmt   
	//   // there is  quantity of real channels
	//const int NChan_fdmt_act = len_sft * header.m_nchan;
	//(*pfdmt) = CFdmtU(
	//	VAlFmin
	//	, VAlFmax
	//	, NChan_fdmt_act
	//	, (*pLenChunk) / len_sft
	//	, pulse_length
	//	, d_max
	//	, len_sft);

	//

	//size_t szBuff_fdmt = pfdmt->calcSizeAuxBuff_fdmt_();

	//checkCudaErrors(hipMalloc(ppAuxBuff_fdmt, szBuff_fdmt));
	//// 4!
	//

	//// 3. memory allocation for fdmt_ones on GPU  ????
	//size_t szBuff_fdmt_output = pfdmt->calc_size_output();

	//checkCudaErrors(hipMalloc((void**)d_parrfdmt_norm, szBuff_fdmt_output));
	////// 6. calculation fdmt ones
	//pfdmt->process_image(nullptr      // on-device input image
	//	, *ppAuxBuff_fdmt
	//	, *d_parrfdmt_norm	// OUTPUT image,
	//	, true);

	//// 3!

	//


	//// 5. memory allocation for the 3 auxillary hipfftComplex  arrays on GPU	
	////hipfftComplex* pffted_rowsignal = NULL; //1	



	//checkCudaErrors(hipMalloc((void**)ppcarrTemp, QUantBlockComplexNumbers * sizeof(hipfftComplex)));

	//checkCudaErrors(hipMalloc((void**)ppcarrCD_Out, QUantBlockComplexNumbers * sizeof(hipfftComplex)));

	//checkCudaErrors(hipMalloc((void**)ppcarrBuff, QUantBlockComplexNumbers * sizeof(hipfftComplex)));
	//// !5
	//
	//// 5. memory allocation for the 2 auxillary arrays on GPU for input and output of FDMT	
	//size_t szInpOut_fdmt = pfdmt->calc_size_output() + pfdmt->calc_size_input();

	//checkCudaErrors(hipMalloc((void**)ppInpOutBuffFdmt, szInpOut_fdmt));

	//// 5!
	//
	//// !4	
	//**ppChunk = CChunk_gpu(
	//	VAlFmin
	//	, VAlFmax
	//	, header.m_npol
	//	, header.m_nchan
	//	, (*pLenChunk)
	//	, len_sft
	//	, 0
	//	, 0
	//	, header.m_nbits
	//	, d_max
	//	, sigma_bound
	//	, length_sum_wnd
	//	, *pfdmt
	//	, pulse_length
	//);
	//
}
//-----------------------------------------------------------------------------------------
void windowization(float* d_fdmt_normalized, const int Rows, const int Cols, const int width, float* parrImage)
{
	for (int i = 0; i < Rows; ++i)
	{
		for (int j = 0; j < Cols; ++j)
		{
			
			float sum = 0.;
			for (int k = 0; k < width; ++k)
			{
				if ((j + k) < Cols)
				{
					sum += d_fdmt_normalized[i * Cols + j + k];
				}
				else
				{
					sum = 0.;
					break;
				}
				
			}
			parrImage[i * Cols + j] = sum / sqrt((float)width);
		}
	}
}
//----------------------------------------------------
__global__
void fdmt_normalization(fdmt_type_* d_arr, fdmt_type_* d_norm, const int len, float* d_pOutArray)
{

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= len)
	{
		return;
	}
	d_pOutArray[idx] = ((float)d_arr[idx]) / sqrtf(((float)d_norm[idx]) + 1.0E-8);

}








