﻿
#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include "npy.hpp"
#include "kernel.cuh"
#include <chrono>
#include "fileInput.h"
#include "DrawImg.h"
#include "Constants.h"
#include "fdmt.h"

using namespace std;
enum TYPE_OF_PROCESSOR
{
	CPU
	, GPU
};

char strInpFolder[] = "..//FDMT_TESTS//2048";
char strPathOutImageNpyFile_gpu[] = "out_image_GPU.npy";
const bool BDIM_512_1024 = true;
TYPE_OF_PROCESSOR PROCESSOR = CPU;


void printDeviceInfo()
{
	int deviceId;
	int numberOfSMs;
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	hipDeviceProp_t deviceProps;
	hipGetDeviceProperties(&deviceProps, deviceId);
	std::string deviceName = deviceProps.name;
	std::cout << "Device Name: " << deviceName << std::endl;
	std::cout << "Number of SM: " << numberOfSMs << std::endl;
}


//---------------------------------------

int main(int argc, char** argv)
{
	printDeviceInfo();
	//--------------------------------------------------------------------------------------------------------------
	//------------------- prepare to work -------------------------------------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------------------

	// initiating input variables
	int iMaxDT = 0;
	int iImRows = 0, iImCols = 0;
	float val_fmax = 0., val_fmin = 0.;
	//int  nchan = iImRows;// 400;
	readDimensions(strInpFolder, &iImRows, &iImCols);
	// initiate pointer to input image
	fdmt_type_* h_parrImage = (fdmt_type_*)malloc(sizeof(fdmt_type_) * iImRows * iImCols);
	int ireturn = downloadInputData_gpu(strInpFolder, &iMaxDT, h_parrImage, &iImRows, &iImCols,
		&val_fmin, &val_fmax);

	fdmt_type_* u_parrImage = NULL;
	fdmt_type_* u_parrImOut = NULL;
	float tsamp = 1.0;
	
	size_t dt_step = 8;
	size_t dt_min = 1;
	FDMT* pfdmt = new FDMT(val_fmin, val_fmax, iImRows, iImCols, tsamp,
		iMaxDT, dt_step, dt_min);
	const size_t IOutImageRows = pfdmt->get_dt_grid_final().size();
	size_t dmt_size = IOutImageRows * iImCols;
	u_parrImOut = (fdmt_type_*)malloc(dmt_size * sizeof(fdmt_type_));
	if (PROCESSOR == GPU)
	{
		hipMalloc(&u_parrImage, sizeof(fdmt_type_) * iImRows * iImCols);
		hipMemcpy(u_parrImage, h_parrImage, sizeof(fdmt_type_) * iImRows * iImCols, hipMemcpyHostToDevice);
		hipMallocManaged(&u_parrImOut, iImCols * iMaxDT * sizeof(fdmt_type_));
		
	}
	else
	{
		u_parrImage = (fdmt_type_*)malloc(sizeof(fdmt_type_) * iImRows * iImCols);
		memcpy(u_parrImage, h_parrImage, sizeof(fdmt_type_) * iImRows * iImCols);
		u_parrImOut = (fdmt_type_*)malloc(sizeof(fdmt_type_) * iMaxDT * iImCols);
		
	}
	free(h_parrImage);
	
	iMaxDT = iMaxDT ;//iMaxDT * 3 +10;//iMaxDT//;	

	//--------------------------------------------------------------------------------------------------------------
	//-------------------- end of prepare ------------------------------------------------------------------------------------------
	//------------------- begin to calculate cuda var -------------------------------------------------------------------------------------------
	//--------------------------------------------------------------------------------------------------------------
	//iImCols = 1 << 18;
	
	
	// 3. calculations		
	int num = 100;
	auto start = std::chrono::high_resolution_clock::now();

	for (int i = 0; i < num; ++i)
	{
		pfdmt->execute(u_parrImage       // on-device input image	
			, iImRows * iImCols
			, u_parrImOut	// OUTPUT image, dim = IMaxDT x IImgcols
			, dmt_size);
		
	}
	
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "PROCESSOR = "<< PROCESSOR<<"  Time taken by function fncFdmtU_cu : " << duration.count() / ((double)num) << " microseconds" << std::endl;

	delete pfdmt;
	// !3
	
	//4. write  output in .npy:IImgcols * IMaxDT * sizeof(int));
	//hipDeviceSynchronize();
	fdmt_type_* parrImOut = (fdmt_type_*)malloc(iImCols * IOutImageRows * sizeof(fdmt_type_));

	if (PROCESSOR == GPU)
	{		
		hipMemcpy(parrImOut, u_parrImOut, iImCols * IOutImageRows * sizeof(fdmt_type_), hipMemcpyDeviceToHost);
	}
	else
	{
		memcpy(parrImOut, u_parrImOut, iImCols * IOutImageRows * sizeof(fdmt_type_));
	}
	

	//std::vector<float> v1(parrImOut, parrImOut + iImCols * iMaxDT);
	std::vector<float> v1(iImCols * IOutImageRows);
	for (int i = 0; i < iImCols * IOutImageRows; ++i)
	{
		v1.at(i) = (float)parrImOut[i];
	}

	std::array<long unsigned, 2> leshape101{ iImCols ,IOutImageRows };

	npy::SaveArrayAsNumpy(strPathOutImageNpyFile_gpu, false, leshape101.size(), leshape101.data(), v1);

	//--------------------------------------------------------------------------------------------------------------
	//-------------------- end of calculations ------------------------------------------------------------------------------------------
	//------------------- begin to draw output image for cuda -------------------------------------------------------------------------------------------

	float flops = 0;
	if (iImRows == 512)
	{
		flops = GFLPS_512;
	}
	else
	{
		if (iImRows == 1024)
		{
			if (BDIM_512_1024)
			{
				flops = GFLPS_512_1024;
			}
			else
			{
				flops = GFLPS_1024;
			}
		}
		else
		{
			flops = GFLPS_2048;
		}
	}

	cout << "GFLP/sec = " << ((double)flops) / ((double)duration.count() / ((double)num)) * 1.0e6 << endl;

	if (PROCESSOR == GPU)
	{
		hipFree(u_parrImage);
		hipFree(u_parrImOut);
	}
	else
	{
		free(u_parrImage);
		free(u_parrImOut);
	}
	free(parrImOut);		
	

	char filename_cpu[] = "image_gpu.png";
	createImg_(argc, argv, v1, IOutImageRows, iImCols, filename_cpu);

	return 0;
}
