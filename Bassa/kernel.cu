#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <math.h>

#include <array>
#include <iostream>
#include <string>

#include <vector>
//#include <unistd.h>
#include <math.h>
#include <time.h>
#include<errno.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
//#include <getopt.h>
#include <hdf5.h>
#include <stdlib.h>
//#include <cstddef>  // For C++
//#include <cstdint>
//#include <cstdlib>



#define HEADERSIZE 4096
#define DMCONSTANT 2.41e-10

#define _USE_MATH_DEFINES
#include <cmath>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

using namespace std;
// Struct for header information
struct header {
    int64_t headersize, buffersize;
    unsigned int nchan, nsamp, nbit, nif, nsub;
    int machine_id, telescope_id, nbeam, ibeam, sumif;
    double tstart, tsamp, fch1, foff, fcen, bwchan;
    double src_raj, src_dej, az_start, za_start;
    char source_name[80], ifstream[8], inpfile[80];
    char* rawfname[4];
};

struct header read_h5_header(char* fname);
void get_channel_chirp(double fcen, double bw, float dm, int nchan, int nbin, int nsub, hipfftComplex* c);
__global__ void transpose_unpadd_and_detect(hipfftComplex* cp1, hipfftComplex* cp2, int nbin, int nchan, int nfft, int nsub, int noverlap, int nsamp, float* fbuf);
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a, float s);
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b);
static __global__ void PointwiseComplexMultiply(hipfftComplex* a, hipfftComplex* b, hipfftComplex* c, int nx, int ny, int l, float scale);
__global__ void unpack_and_padd(char* dbuf0, char* dbuf1, char* dbuf2, char* dbuf3, int nsamp, int nbin, int nfft, int nsub, int noverlap, hipfftComplex* cp1, hipfftComplex* cp2);
__global__ void swap_spectrum_halves(hipfftComplex* cp1, hipfftComplex* cp2, int nx, int ny);
__global__ void compute_chirp(double fcen, double bw, float* dm, int nchan, int nbin, int nsub, int ndm, hipfftComplex* c);
__global__ void compute_block_sums(float* z, int nchan, int nblock, int nsum, float* bs1, float* bs2);
__global__ void compute_channel_statistics(int nchan, int nblock, int nsum, float* bs1, float* bs2, float* zavg, float* zstd);
__global__ void redigitize(float* z, int nchan, int nblock, int nsum, float* zavg, float* zstd, float zmin, float zmax, unsigned char* cz);
__global__ void decimate_and_redigitize(float* z, int ndec, int nchan, int nblock, int nsum, float* zavg, float* zstd, float zmin, float zmax, unsigned char* cz);
void write_filterbank_header(struct header h, FILE* file);
__global__ void kernel_Sum_cols(unsigned char* d_inp, const int NRows, const int NCols, int* d_arrOut);
int readHeader5InBinary(const char* strFilePath, header& Head);
int writeOptions(const char* strFilePath, const int noverlap, const int nbin, const int ndec, const char* obsid
    , const int part, const int device);
int readOptions(const char* strFilePath, int& noverlap, int& nbin, int& ndec, char* obsid
    , int& part, int& device);

// Usage
void usage()
{
    printf("cdmt -P <part> -d <DM start,step,num> -D <GPU device> -b <ndec> -N <forward FFT size> -n <overlap region> -o <outputname> <file.h5>\n\n");
    printf("Compute coherently dedispersed SIGPROC filterbank files from LOFAR complex voltage data in HDF5 format.\n");
    printf("-P <part>        Specify part number for input file [integer, default: 0]\n");
    printf("-D <GPU device>  Select GPU device [integer, default: 0]\n");
    printf("-b <ndec>        Number of time samples to average [integer, default: 1]\n");
    printf("-d <DM start, step, num>  DM start and stepsize, number of DM trials\n");
    printf("-o <outputname>           Output filename [default: cdmt]\n");
    printf("-N <forward FFT size>     Forward FFT size [integer, default: 65536]\n");
    printf("-n <overlap region>       Overlap region [integer, default: 2048]\n");

    return;
}

int main(int argc, char* argv[])
{
    int i, nsamp, nfft, mbin, nvalid, nchan = 8, nbin = 65536, noverlap = 2048, nsub = 20, ndm, ndec = 1;
    int idm, iblock, nread, mchan, msamp, mblock, msum = 1024;
    char* header, * h5buf[4], * dh5buf[4];
    FILE* rawfile[4], * file;
    unsigned char* cbuf, * dcbuf;
    float* fbuf, * dfbuf;
    float* bs1, * bs2, * zavg, * zstd;
    hipfftComplex* cp1, * cp2, * dc, * cp1p, * cp2p;
    hipfftHandle ftc2cf, ftc2cb;
    int idist, odist, iembed, oembed, istride, ostride;
    dim3 blocksize, gridsize;
    struct header h5;
    clock_t startclock;
    float* dm, * ddm, dm_start, dm_step;
    char fname[128], fheader[1024], * h5fname, obsid[128] = "cdmt";
    int bytes_read;
    int part = 0, device = 0;
    int arg = 0;
    FILE** outfile;
    /*
    // Read options
    if (argc > 1) {
        while ((arg = getopt(argc, argv, "P:d:D:ho:b:N:n:")) != -1) {
            switch (arg) {

            case 'n':
                noverlap = atoi(optarg);
                break;

            case 'N':
                nbin = atoi(optarg);
                break;

            case 'b':
                ndec = atoi(optarg);
                break;

            case 'o':
                strcpy(obsid, optarg);
                break;

            case 'P':
                part = atoi(optarg);
                break;

            case 'D':
                device = atoi(optarg);
                break;

            case 'd':
                sscanf(optarg, "%f,%f,%d", &dm_start, &dm_step, &ndm);
                break;

            case 'h':
                usage();
                return 0;
            }
        }
    }
    else {
        usage();
        return 0;
    }
    h5fname = argv[optind];
    */
    // Read HDF5 header
    //h5 = read_h5_header(h5fname); !!
    //readOptions("Options.bin", noverlap, nbin, ndec, obsid, part, device);
    // cdmt -n 8192 -N 32768 -P 001 -b 4 -d 0.5,1.0,80 -o tmp/test L2012176_SAP000_B000_S0_P001_bf.h5
    //

    h5fname = new char[200];
    memset(h5fname, 0, 200 * sizeof(char));
    strcpy(h5fname, "D://BASSA//hdf5_data//L2012176_SAP000_B000_S0_P001_bf.h5");
    h5 = read_h5_header(h5fname);
    dm_start = 0.5;
    dm_step = 1.0;
    ndm = 80;
    noverlap = 8192;
    nbin = 32768;
    ndec = 4;
    part = 0;
    device = 0;
    strcpy(obsid, "tmp/test");
    //readHeader5InBinary("Header5File.bin", h5);

    // Set number of subbands
    nsub = h5.nsub;

    // Adjust header for filterbank format
    h5.tsamp *= nchan * ndec;
    h5.nchan = nsub * nchan;
    h5.nbit = 8;
    h5.fch1 = h5.fcen + 0.5 * h5.nsub * h5.bwchan - 0.5 * h5.bwchan / nchan;
    h5.foff = -fabs(h5.bwchan / nchan);

    // Data size
    nvalid = nbin - 2 * noverlap;
    nsamp = 100 * nvalid;
    nfft = (int)ceil(nsamp / (float)nvalid);
    mbin = nbin / nchan;
    mchan = nsub * nchan;
    msamp = nsamp / nchan;
    mblock = msamp / msum;

    printf("nbin: %d nfft: %d nsub: %d mbin: %d nchan: %d nsamp: %d nvalid: %d\n", nbin, nfft, nsub, mbin, nchan, nsamp, nvalid);

    // Set device
    checkCudaErrors(hipSetDevice(device));

    // Allocate memory for complex timeseries
    checkCudaErrors(hipMalloc((void**)&cp1, sizeof(hipfftComplex) * nbin * nfft * nsub));
    checkCudaErrors(hipMalloc((void**)&cp2, sizeof(hipfftComplex) * nbin * nfft * nsub));
    checkCudaErrors(hipMalloc((void**)&cp1p, sizeof(hipfftComplex) * nbin * nfft * nsub));
    checkCudaErrors(hipMalloc((void**)&cp2p, sizeof(hipfftComplex) * nbin * nfft * nsub));

    // Allocate device memory for chirp
    checkCudaErrors(hipMalloc((void**)&dc, sizeof(hipfftComplex) * nbin * nsub * ndm));

    // Allocate device memory for block sums
    checkCudaErrors(hipMalloc((void**)&bs1, sizeof(float) * mblock * mchan));
    checkCudaErrors(hipMalloc((void**)&bs2, sizeof(float) * mblock * mchan));

    // Allocate device memory for channel averages and standard deviations
    checkCudaErrors(hipMalloc((void**)&zavg, sizeof(float) * mchan));
    checkCudaErrors(hipMalloc((void**)&zstd, sizeof(float) * mchan));

    // Allocate memory for redigitized output and header
    header = (char*)malloc(sizeof(char) * HEADERSIZE);
    for (i = 0; i < 4; i++) {
        h5buf[i] = (char*)malloc(sizeof(char) * nsamp * nsub);
        checkCudaErrors(hipMalloc((void**)&dh5buf[i], sizeof(char) * nsamp * nsub));
    }

    // Allocate output buffers
    fbuf = (float*)malloc(sizeof(float) * nsamp * nsub);
    checkCudaErrors(hipMalloc((void**)&dfbuf, sizeof(float) * nsamp * nsub));
    cbuf = (unsigned char*)malloc(sizeof(unsigned char) * msamp * mchan / ndec);
    checkCudaErrors(hipMalloc((void**)&dcbuf, sizeof(unsigned char) * msamp * mchan / ndec));

    // Allocate DMs and copy to device
    dm = (float*)malloc(sizeof(float) * ndm);
    for (idm = 0; idm < ndm; idm++)
        dm[idm] = dm_start + (float)idm * dm_step;
    checkCudaErrors(hipMalloc((void**)&ddm, sizeof(float) * ndm));
    checkCudaErrors(hipMemcpy(ddm, dm, sizeof(float) * ndm, hipMemcpyHostToDevice));

    // Generate FFT plan (batch in-place forward FFT)
    idist = nbin;  odist = nbin;  iembed = nbin;  oembed = nbin;  istride = 1;  ostride = 1;
    checkCudaErrors(hipfftPlanMany(&ftc2cf, 1, &nbin, &iembed, istride, idist, &oembed, ostride, odist, HIPFFT_C2C, nfft * nsub));

    // Generate FFT plan (batch in-place backward FFT)
    idist = mbin;  odist = mbin;  iembed = mbin;  oembed = mbin;  istride = 1;  ostride = 1;
    checkCudaErrors(hipfftPlanMany(&ftc2cb, 1, &mbin, &iembed, istride, idist, &oembed, ostride, odist, HIPFFT_C2C, nchan * nfft * nsub));

    // Compute chirp
    blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
    gridsize.x = nsub / blocksize.x + 1; gridsize.y = nchan / blocksize.y + 1; gridsize.z = ndm / blocksize.z + 1;
    compute_chirp << <gridsize, blocksize >> > (h5.fcen, nsub * h5.bwchan, ddm, nchan, nbin, nsub, ndm, dc);

    // Write temporary filterbank header
    FILE*file0 = fopen("tmp/header.fil", "w");// ОТЛИЧИЕ В ОТКРЫТИИ ФАЙЛА!!
    write_filterbank_header(h5, file0);
    fclose(file0);
    file = fopen("tmp/header.fil", "r");
    bytes_read = fread(fheader, sizeof(char), 1024, file);
    fclose(file);

    // Format file names and open
    outfile = (FILE**)malloc(sizeof(FILE*) * ndm);
    for (idm = 0; idm < ndm; idm++) {
        sprintf(fname, "%s_cDM%06.2f_P%03d.fil", obsid, dm[idm], part);

        outfile[idm] = fopen(fname, "w");
    }

    // Write headers
    for (idm = 0; idm < ndm; idm++) {
        // Send header
        fwrite(fheader, sizeof(char), bytes_read, outfile[idm]);
    }

    // Read files
    for (i = 0; i < 4; i++) {
        rawfile[i] = fopen(h5.rawfname[i], "rb");
        
    }

    // Loop over input file contents
    for (iblock = 0;; iblock++)
    {
        // Read block
        startclock = clock();
        for (i = 0; i < 4; i++)
        {
            //nread = fread(h5buf[i], sizeof(char), nsamp * nsub, rawfile[i]) / nsub;
            nread = fread(h5buf[i], sizeof(char), nsamp * nsub, rawfile[i]);
            nread = nread / nsub;
            std::cout << "nread =" << nread << std::endl;
        }
        if (nread == 0)
            break;
        printf("Block: %d: Read %d MB in %.2f s\n", iblock, sizeof(char) * nread * nsub * 4 / (1 << 20), (float)(clock() - startclock) / CLOCKS_PER_SEC);

        // Copy buffers to device
        startclock = clock();
        for (i = 0; i < 4; i++)
        {
            checkCudaErrors(hipMemcpy(dh5buf[i], h5buf[i], sizeof(char) * nread * nsub, hipMemcpyHostToDevice));
        }
        // Unpack data and padd data
        blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
        gridsize.x = nbin / blocksize.x + 1; gridsize.y = nfft / blocksize.y + 1; gridsize.z = nsub / blocksize.z + 1;
        unpack_and_padd << <gridsize, blocksize >> > (dh5buf[0], dh5buf[1], dh5buf[2], dh5buf[3], nread, nbin, nfft, nsub, noverlap, cp1p, cp2p);

        // Perform FFTs
        checkCudaErrors(hipfftExecC2C(ftc2cf, (hipfftComplex*)cp1p, (hipfftComplex*)cp1p, HIPFFT_FORWARD));
        checkCudaErrors(hipfftExecC2C(ftc2cf, (hipfftComplex*)cp2p, (hipfftComplex*)cp2p, HIPFFT_FORWARD));

        // Swap spectrum halves for large FFTs
        blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
        gridsize.x = nbin / blocksize.x + 1; gridsize.y = nfft * nsub / blocksize.y + 1; gridsize.z = 1;
        swap_spectrum_halves << <gridsize, blocksize >> > (cp1p, cp2p, nbin, nfft * nsub);

        // Loop over dms
        for (idm = 0; idm < ndm; idm++) {

            // Perform complex multiplication of FFT'ed data with chirp
            blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
            gridsize.x = nbin * nsub / blocksize.x + 1; gridsize.y = nfft / blocksize.y + 1; gridsize.z = 1;
            PointwiseComplexMultiply << <gridsize, blocksize >> > (cp1p, dc, cp1, nbin * nsub, nfft, idm, 1.0 / (float)nbin);
            PointwiseComplexMultiply << <gridsize, blocksize >> > (cp2p, dc, cp2, nbin * nsub, nfft, idm, 1.0 / (float)nbin);

            // Swap spectrum halves for small FFTs
            blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
            gridsize.x = mbin / blocksize.x + 1; gridsize.y = nchan * nfft * nsub / blocksize.y + 1; gridsize.z = 1;
            swap_spectrum_halves << <gridsize, blocksize >> > (cp1, cp2, mbin, nchan * nfft * nsub);

            // Perform FFTs
            checkCudaErrors(hipfftExecC2C(ftc2cb, (hipfftComplex*)cp1, (hipfftComplex*)cp1, HIPFFT_BACKWARD));
            checkCudaErrors(hipfftExecC2C(ftc2cb, (hipfftComplex*)cp2, (hipfftComplex*)cp2, HIPFFT_BACKWARD));

            // Detect data
            blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
            gridsize.x = mbin / blocksize.x + 1; gridsize.y = nchan / blocksize.y + 1; gridsize.z = nfft / blocksize.z + 1;
            transpose_unpadd_and_detect << <gridsize, blocksize >> > (cp1, cp2, mbin, nchan, nfft, nsub, noverlap / nchan, nread / nchan, dfbuf);

            // Compute block sums for redigitization
            blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
            gridsize.x = mchan / blocksize.x + 1; gridsize.y = mblock / blocksize.y + 1; gridsize.z = 1;
            compute_block_sums << <gridsize, blocksize >> > (dfbuf, mchan, mblock, msum, bs1, bs2);

            // Compute channel stats
            blocksize.x = 32; blocksize.y = 1; blocksize.z = 1;
            gridsize.x = mchan / blocksize.x + 1; gridsize.y = 1; gridsize.z = 1;
            compute_channel_statistics << <gridsize, blocksize >> > (mchan, mblock, msum, bs1, bs2, zavg, zstd);

            // Redigitize data to 8bits
            blocksize.x = 32; blocksize.y = 32; blocksize.z = 1;
            gridsize.x = mchan / blocksize.x + 1; gridsize.y = mblock / blocksize.y + 1; gridsize.z = 1;
            if (ndec == 1)
                redigitize << <gridsize, blocksize >> > (dfbuf, mchan, mblock, msum, zavg, zstd, 3.0, 5.0, dcbuf);
            else
                decimate_and_redigitize << <gridsize, blocksize >> > (dfbuf, ndec, mchan, mblock, msum, zavg, zstd, 3.0, 5.0, dcbuf);

            //********************************************************************************************
            //********************************************************************************************
            //********************************************************************************************
            int* dcbuf_arr = NULL;
            checkCudaErrors(hipMalloc((void**)&dcbuf_arr, sizeof(int) * mchan));
            int blocksPerGrid = mchan;
            unsigned int treadsPerBlock = 256;
            std::size_t sz = treadsPerBlock * sizeof(int);
            kernel_Sum_cols << < blocksPerGrid, treadsPerBlock, sz >> > (dcbuf, mchan, msamp / ndec, dcbuf_arr);
            checkCudaErrors(hipMemcpy(cbuf, dcbuf, sizeof(int) * mchan, hipMemcpyDeviceToHost));
            fwrite(cbuf, sizeof(int), mchan, outfile[idm]);
            hipFree(dcbuf_arr);
            //********************************************************************************************
            //********************************************************************************************
            //********************************************************************************************

            //// Copy buffer to host
            //checkCudaErrors(hipMemcpy(cbuf,dcbuf,sizeof(unsigned char)*msamp*mchan/ndec,hipMemcpyDeviceToHost));

            //// Write buffer
            //fwrite(cbuf,sizeof(char),nread*nsub/ndec,outfile[idm]);
        }
        printf("Processed %d DMs in %.2f s\n", ndm, (float)(clock() - startclock) / CLOCKS_PER_SEC);
    }

    // Close files
    for (i = 0; i < ndm; i++)
        fclose(outfile[i]);

    // Close files
    for (i = 0; i < 4; i++)
        fclose(rawfile[i]);

    // Free
    free(header);
    for (i = 0; i < 4; i++) {
        free(h5buf[i]);
        hipFree(dh5buf);
        free(h5.rawfname[i]);
    }
    free(fbuf);
    free(dm);
    free(cbuf);
    free(outfile);

    hipFree(dfbuf);
    hipFree(dcbuf);
    hipFree(cp1);
    hipFree(cp2);
    hipFree(cp1p);
    hipFree(cp2p);
    hipFree(dc);
    hipFree(bs1);
    hipFree(bs2);
    hipFree(zavg);
    hipFree(zstd);
    hipFree(ddm);

    // Free plan
    hipfftDestroy(ftc2cf);
    hipfftDestroy(ftc2cb);

    return 0;
}
// !!!!!!
// This is a simple H5 reader for complex voltage data. Very little
// error checking is done.
struct header read_h5_header(char* fname)
{
    int i, len, ibeam, isap;
    struct header h;
    hid_t file_id, attr_id, sap_id, beam_id, memtype, group_id, space, coord_id;
    char* string, * pch;
    const char* stokes[] = { "_S0_","_S1_","_S2_","_S3_" };
    char* froot, * fpart, * ftest, group[32];
    FILE* file;

    // Find filenames
    for (i = 0; i < 4; i++) {
        pch = strstr(fname, stokes[i]);
        if (pch != NULL)
            break;
    }
    len = strlen(fname) - strlen(pch);
    froot = (char*)malloc(sizeof(char) * (len + 1));
    memset(froot, 0, sizeof(char) * (len + 1));
    fpart = (char*)malloc(sizeof(char) * (strlen(pch) - 6));
    memset(fpart, 0, sizeof(char) * (strlen(pch) - 6));
    ftest = (char*)malloc(sizeof(char) * (len + 20));
    memset(ftest,0, sizeof(char) * (len + 20));
    strncpy(froot, fname, len);
    strncpy(fpart, pch + 4, strlen(pch) - 7);

    // Check files
    for (i = 0; i < 4; i++) {
        // Format file name
        sprintf(ftest, "%s_S%d_%s.raw", froot, i, fpart);
        // Try to open
        if ((file = fopen(ftest, "r")) != NULL) {
            fclose(file);
        }
        else {
            fprintf(stderr, "Raw file %s not found\n", ftest);
            exit(-1);
        }
        h.rawfname[i] = (char*)malloc(sizeof(char) * (strlen(ftest) + 1));
        strcpy(h.rawfname[i], ftest);
    }

    // Get beam number
    for (i = 0; i < 4; i++) {
        pch = strstr(fname, "_B");
        if (pch != NULL)
            break;
    }
    sscanf(pch + 2, "%d", &ibeam);

    // Get SAP number
    for (i = 0; i < 4; i++) {
        pch = strstr(fname, "_SAP");
        if (pch != NULL)
            break;
    }
    sscanf(pch + 4, "%d", &isap);

    // Free
    free(froot);
    free(fpart);
    free(ftest);

    // Open file
    file_id = H5Fopen(fname, H5F_ACC_RDONLY, H5P_DEFAULT);

    // Open subarray pointing group
    sprintf(group, "SUB_ARRAY_POINTING_%03d", isap);
    sap_id = H5Gopen(file_id, group, H5P_DEFAULT);

    // Start MJD
    attr_id = H5Aopen(sap_id, "EXPTIME_START_MJD", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.tstart);
    H5Aclose(attr_id);

    // Declination
    attr_id = H5Aopen(sap_id, "POINT_DEC", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.src_dej);
    H5Aclose(attr_id);

    // Right ascension
    attr_id = H5Aopen(sap_id, "POINT_RA", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.src_raj);
    H5Aclose(attr_id);

    // Open beam
    sprintf(group, "BEAM_%03d", ibeam);
    beam_id = H5Gopen(sap_id, group, H5P_DEFAULT);

    // Number of samples
    attr_id = H5Aopen(beam_id, "NOF_SAMPLES", H5P_DEFAULT);
    H5Aread(attr_id, H5T_STD_U32LE, &h.nsamp);
    H5Aclose(attr_id);

    // Center frequency
    attr_id = H5Aopen(beam_id, "BEAM_FREQUENCY_CENTER", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.fcen);
    H5Aclose(attr_id);

    // Center frequency unit
    attr_id = H5Aopen(beam_id, "BEAM_FREQUENCY_CENTER_UNIT", H5P_DEFAULT);
    memtype = H5Tcopy(H5T_C_S1);
    H5Tset_size(memtype, H5T_VARIABLE);
    H5Aread(attr_id, memtype, &string);
    H5Aclose(attr_id);
    if (strcmp(string, "Hz") == 0)
        h.fcen /= 1e6;

    // Channel bandwidth
    attr_id = H5Aopen(beam_id, "CHANNEL_WIDTH", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.bwchan);
    H5Aclose(attr_id);

    // Center frequency unit
    attr_id = H5Aopen(beam_id, "CHANNEL_WIDTH_UNIT", H5P_DEFAULT);
    memtype = H5Tcopy(H5T_C_S1);
    H5Tset_size(memtype, H5T_VARIABLE);
    H5Aread(attr_id, memtype, &string);
    H5Aclose(attr_id);
    if (strcmp(string, "Hz") == 0)
        h.bwchan /= 1e6;

    // Get source
    attr_id = H5Aopen(beam_id, "TARGETS", H5P_DEFAULT);
    memtype = H5Tcopy(H5T_C_S1);
    H5Tset_size(memtype, H5T_VARIABLE);
    H5Aread(attr_id, memtype, &string);
    H5Aclose(attr_id);
    strcpy(h.source_name, string);

    // Open coordinates
    coord_id = H5Gopen(beam_id, "COORDINATES", H5P_DEFAULT);

    // Open coordinate 0
    group_id = H5Gopen(coord_id, "COORDINATE_0", H5P_DEFAULT);

    // Sampling time
    attr_id = H5Aopen(group_id, "INCREMENT", H5P_DEFAULT);
    H5Aread(attr_id, H5T_IEEE_F64LE, &h.tsamp);
    H5Aclose(attr_id);

    // Close group
    H5Gclose(group_id);

    // Open coordinate 1
    group_id = H5Gopen(coord_id, "COORDINATE_1", H5P_DEFAULT);

    // Number of subbands
    attr_id = H5Aopen(group_id, "AXIS_VALUES_WORLD", H5P_DEFAULT);
    space = H5Aget_space(attr_id);
    h.nsub = H5Sget_simple_extent_npoints(space);
    H5Aclose(attr_id);

    // Close group
    H5Gclose(group_id);

    // Close coordinates
    H5Gclose(coord_id);

    // Close beam, sap and file
    H5Gclose(beam_id);
    H5Gclose(sap_id);
    H5Fclose(file_id);

    return h;
}

// Scale hipfftComplex 
static __device__ __host__ inline hipfftComplex ComplexScale(hipfftComplex a, float s)
{
    hipfftComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Pointwise complex multiplication (and scaling)
static __global__ void PointwiseComplexMultiply(hipfftComplex* a, hipfftComplex* b, hipfftComplex* c, int nx, int ny, int l, float scale)
{
    int i, j, k;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        k = i + nx * j;
        c[k] = ComplexScale(ComplexMul(a[k], b[i + nx * l]), scale);
    }
}

// Compute chirp
__global__ void compute_chirp(double fcen, double bw, float* dm, int nchan, int nbin, int nsub, int ndm, hipfftComplex* c)
{
    int ibin, ichan, isub, idm, mbin, idx;
    double s, rt, t, f, fsub, fchan, bwchan, bwsub;

    // Number of channels per subband
    mbin = nbin / nchan;

    // Subband bandwidth
    bwsub = bw / nsub;

    // Channel bandwidth
    bwchan = bw / (nchan * nsub);

    // Indices of input data
    isub = blockIdx.x * blockDim.x + threadIdx.x;
    ichan = blockIdx.y * blockDim.y + threadIdx.y;
    idm = blockIdx.z * blockDim.z + threadIdx.z;

    // Keep in range
    if (isub < nsub && ichan < nchan && idm < ndm) {
        // Main constant
        s = 2.0 * M_PI * dm[idm] / DMCONSTANT;

        // Frequencies
        fsub = fcen - 0.5 * bw + bw * (float)isub / (float)nsub + 0.5 * bw / (float)nsub;
        fchan = fsub - 0.5 * bwsub + bwsub * (float)ichan / (float)nchan + 0.5 * bwsub / (float)nchan;

        // Loop over bins in channel
        for (ibin = 0; ibin < mbin; ibin++) {
            // Bin frequency
            f = -0.5 * bwchan + bwchan * (float)ibin / (float)mbin + 0.5 * bwchan / (float)mbin;

            // Phase delay
            rt = -f * f * s / ((fchan + f) * fchan * fchan);

            // Taper
            t = 1.0 / sqrt(1.0 + pow((f / (0.47 * bwchan)), 80));

            // Index
            idx = ibin + ichan * mbin + isub * mbin * nchan + idm * nsub * mbin * nchan;

            // Chirp
            c[idx].x = cos(rt) * t;
            c[idx].y = sin(rt) * t;
        }
    }

    return;
}

// Unpack the input buffer and generate complex timeseries. The output
// timeseries are padded with noverlap samples on either side for the
// convolution.
__global__ void unpack_and_padd(char* dbuf0, char* dbuf1, char* dbuf2, char* dbuf3, int nsamp, int nbin, int nfft, int nsub, int noverlap, hipfftComplex* cp1, hipfftComplex* cp2)
{
    int64_t ibin, ifft, isamp, isub, idx1, idx2;

    // Indices of input data
    ibin = blockIdx.x * blockDim.x + threadIdx.x;
    ifft = blockIdx.y * blockDim.y + threadIdx.y;
    isub = blockIdx.z * blockDim.z + threadIdx.z;

    // Only compute valid threads
    if (ibin < nbin && ifft < nfft && isub < nsub) {
        idx1 = ibin + nbin * isub + nsub * nbin * ifft;
        isamp = ibin + (nbin - 2 * noverlap) * ifft - noverlap;
        idx2 = isub + nsub * isamp;
        if (isamp < 0 || isamp >= nsamp) {
            cp1[idx1].x = 0.0;
            cp1[idx1].y = 0.0;
            cp2[idx1].x = 0.0;
            cp2[idx1].y = 0.0;
        }
        else {
            cp1[idx1].x = (float)dbuf0[idx2];
            cp1[idx1].y = (float)dbuf1[idx2];
            cp2[idx1].x = (float)dbuf2[idx2];
            cp2[idx1].y = (float)dbuf3[idx2];
        }
    }

    return;
}

// Since complex-to-complex FFTs put the center frequency at bin zero
// in the frequency domain, the two halves of the spectrum need to be
// swapped.
__global__ void swap_spectrum_halves(hipfftComplex* cp1, hipfftComplex* cp2, int nx, int ny)
{
    int64_t i, j, k, l, m;
    hipfftComplex tp1, tp2;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx / 2 && j < ny) {
        if (i < nx / 2)
            k = i + nx / 2;
        else
            k = i - nx / 2;
        l = i + nx * j;
        m = k + nx * j;
        tp1.x = cp1[l].x;
        tp1.y = cp1[l].y;
        tp2.x = cp2[l].x;
        tp2.y = cp2[l].y;
        cp1[l].x = cp1[m].x;
        cp1[l].y = cp1[m].y;
        cp2[l].x = cp2[m].x;
        cp2[l].y = cp2[m].y;
        cp1[m].x = tp1.x;
        cp1[m].y = tp1.y;
        cp2[m].x = tp2.x;
        cp2[m].y = tp2.y;
    }

    return;
}

// After the segmented FFT the data is in a cube of nbin by nchan by
// nfft, where nbin and nfft are the time indices. Here we rearrange
// the 3D data cube into a 2D array of frequency and time, while also
// removing the overlap regions and detecting (generating Stokes I).
__global__ void transpose_unpadd_and_detect(hipfftComplex* cp1, hipfftComplex* cp2, int nbin, int nchan, int nfft, int nsub, int noverlap, int nsamp, float* fbuf)
{
    int64_t ibin, ichan, ifft, isub, isamp, idx1, idx2;

    ibin = blockIdx.x * blockDim.x + threadIdx.x;
    ichan = blockIdx.y * blockDim.y + threadIdx.y;
    ifft = blockIdx.z * blockDim.z + threadIdx.z;
    if (ibin < nbin && ichan < nchan && ifft < nfft) {
        // Loop over subbands
        for (isub = 0; isub < nsub; isub++) {
            // Padded array index
            //      idx1=ibin+nbin*isub+nsub*nbin*(ichan+nchan*ifft);
            idx1 = ibin + ichan * nbin + (nsub - isub - 1) * nbin * nchan + ifft * nbin * nchan * nsub;

            // Time index
            isamp = ibin + (nbin - 2 * noverlap) * ifft - noverlap;

            // Output array index
            idx2 = (nchan - ichan - 1) + isub * nchan + nsub * nchan * isamp;

            // Select data points from valid region
            if (ibin >= noverlap && ibin <= nbin - noverlap && isamp >= 0 && isamp < nsamp)
                fbuf[idx2] = cp1[idx1].x * cp1[idx1].x + cp1[idx1].y * cp1[idx1].y + cp2[idx1].x * cp2[idx1].x + cp2[idx1].y * cp2[idx1].y;
        }
    }

    return;
}

void send_string(const char* string, FILE* file)
{
    int len;

    len = strlen(string);
    fwrite(&len, sizeof(int), 1, file);
    fwrite(string, sizeof(char), len, file);

    return;
}

void send_float(const char* string, float x, FILE* file)
{
    send_string(string, file);
    fwrite(&x, sizeof(float), 1, file);

    return;
}

void send_int(const char* string, int x, FILE* file)
{
    send_string(string, file);
    fwrite(&x, sizeof(int), 1, file);

    return;
}

void send_double(const char* string, double x, FILE* file)
{
    send_string(string, file);
    fwrite(&x, sizeof(double), 1, file);

    return;
}

double dec2sex(double x)
{
    double d, sec, min, deg;
    char sign;
    char tmp[32];

    sign = (x < 0 ? '-' : ' ');
    x = 3600.0 * fabs(x);

    sec = fmod(x, 60.0);
    x = (x - sec) / 60.0;
    min = fmod(x, 60.0);
    x = (x - min) / 60.0;
    deg = x;

    sprintf(tmp, "%c%02d%02d%09.6lf", sign, (int)deg, (int)min, sec);
    sscanf(tmp, "%lf", &d);

    return d;
}

void write_filterbank_header(struct header h, FILE* file)
{
    double ra, de;


    ra = dec2sex(h.src_raj / 15.0);
    de = dec2sex(h.src_dej);

    send_string("HEADER_START", file);
    send_string("rawdatafile", file);
    send_string(h.rawfname[0], file);
    send_string("source_name", file);
    send_string(h.source_name, file);
    send_int("machine_id", 11, file);
    send_int("telescope_id", 11, file);
    send_double("src_raj", ra, file);
    send_double("src_dej", de, file);
    send_int("data_type", 1, file);
    send_double("fch1", h.fch1, file);
    send_double("foff", h.foff, file);
    send_int("nchans", h.nchan, file);
    send_int("nbeams", 0, file);
    send_int("ibeam", 0, file);
    send_int("nbits", h.nbit, file);
    send_double("tstart", h.tstart, file);
    send_double("tsamp", h.tsamp, file);
    send_int("nifs", 1, file);
    send_string("HEADER_END", file);

    return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_block_sums(float* z, int nchan, int nblock, int nsum, float* bs1, float* bs2)
{
    int64_t ichan, iblock, isum, idx1, idx2;

    ichan = blockIdx.x * blockDim.x + threadIdx.x;
    iblock = blockIdx.y * blockDim.y + threadIdx.y;
    if (ichan < nchan && iblock < nblock) {
        idx1 = ichan + nchan * iblock;
        bs1[idx1] = 0.0;
        bs2[idx1] = 0.0;
        for (isum = 0; isum < nsum; isum++) {
            idx2 = ichan + nchan * (isum + iblock * nsum);
            bs1[idx1] += z[idx2];
            bs2[idx1] += z[idx2] * z[idx2];
        }
    }

    return;
}

// Compute segmented sums for later computation of offset and scale
__global__ void compute_channel_statistics(int nchan, int nblock, int nsum, float* bs1, float* bs2, float* zavg, float* zstd)
{
    int64_t ichan, iblock, idx1;
    double s1, s2;

    ichan = blockIdx.x * blockDim.x + threadIdx.x;
    if (ichan < nchan) {
        s1 = 0.0;
        s2 = 0.0;
        for (iblock = 0; iblock < nblock; iblock++) {
            idx1 = ichan + nchan * iblock;
            s1 += bs1[idx1];
            s2 += bs2[idx1];
        }
        zavg[ichan] = s1 / (float)(nblock * nsum);
        zstd[ichan] = s2 / (float)(nblock * nsum) - zavg[ichan] * zavg[ichan];
        zstd[ichan] = sqrt(zstd[ichan]);
    }

    return;
}

// Redigitize the filterbank to 8 bits in segments
__global__ void redigitize(float* z, int nchan, int nblock, int nsum, float* zavg, float* zstd, float zmin, float zmax, unsigned char* cz)
{
    int64_t ichan, iblock, isum, idx1;
    float zoffset, zscale;

    ichan = blockIdx.x * blockDim.x + threadIdx.x;
    iblock = blockIdx.y * blockDim.y + threadIdx.y;
    if (ichan < nchan && iblock < nblock) {
        zoffset = zavg[ichan] - zmin * zstd[ichan];
        zscale = (zmin + zmax) * zstd[ichan];

        for (isum = 0; isum < nsum; isum++) {
            idx1 = ichan + nchan * (isum + iblock * nsum);
            z[idx1] -= zoffset;
            z[idx1] *= 256.0 / zscale;
            cz[idx1] = (unsigned char)z[idx1];
            if (z[idx1] < 0.0) cz[idx1] = 0;
            if (z[idx1] > 255.0) cz[idx1] = 255;
        }
    }

    return;
}

// Decimate and Redigitize the filterbank to 8 bits in segments
__global__ void decimate_and_redigitize(float* z, int ndec, int nchan, int nblock, int nsum, float* zavg, float* zstd, float zmin, float zmax, unsigned char* cz)
{
    int64_t ichan, iblock, isum, idx1, idx2, idec;
    float zoffset, zscale, ztmp;

    ichan = blockIdx.x * blockDim.x + threadIdx.x;
    iblock = blockIdx.y * blockDim.y + threadIdx.y;
    if (ichan < nchan && iblock < nblock) {
        zoffset = zavg[ichan] - zmin * zstd[ichan];
        zscale = (zmin + zmax) * zstd[ichan];

        for (isum = 0; isum < nsum; isum += ndec) {
            idx2 = ichan + nchan * (isum / ndec + iblock * nsum / ndec);
            for (idec = 0, ztmp = 0.0; idec < ndec; idec++) {
                idx1 = ichan + nchan * (isum + idec + iblock * nsum);
                ztmp += z[idx1];
            }
            ztmp /= (float)ndec;
            ztmp -= zoffset;
            ztmp *= 256.0 / zscale;
            cz[idx2] = (unsigned char)ztmp;
            if (ztmp < 0.0) cz[idx2] = 0;
            if (ztmp > 255.0) cz[idx2] = 255;
        }
    }

    return;
}

//-----------------------------------------------------------------
__global__ void kernel_Sum_cols(unsigned char* d_inp, const int NRows, const int NCols, int* d_arrOut)
{
    extern __shared__ int sdata[];

    unsigned char* pTemp = d_inp + NCols * blockIdx.x + threadIdx.x;

    unsigned int tid = threadIdx.x;
    unsigned int i = tid;
    if (tid >= NCols)
    {
        sdata[tid] = 0;

    }
    else
    {
        int localSum = 0;



        while (i < NCols)
        {
            localSum += (int)(pTemp[i]);
            pTemp += blockDim.x;
            i += blockDim.x;

        }


    }
    __syncthreads();

    // Parallel reduction within the block to sum partial sums
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }

    // Only thread 0 within each block computes the block's sum
    if (tid == 0)
    {
        d_arrOut[blockIdx.x] = sdata[0];

    }
    __syncthreads();

}
//---------------------------------------------------
int readHeader5InBinary(const char* strFilePath, header& Head)
{

    // Open file for reading in binary mode
    FILE* file = fopen(strFilePath, "rb");

    if (file == NULL) {
        perror("Error opening file");
        return 1;
    }



    // Write the variables to the file

    fread(&Head.headersize, sizeof(int64_t), 1, file);
    fread(&Head.buffersize, sizeof(int64_t), 1, file);//
    fread(&Head.nchan, sizeof(unsigned int), 1, file);
    fread(&Head.nsamp, sizeof(unsigned int), 1, file);
    fread(&Head.nbit, sizeof(unsigned int), 1, file);
    fread(&Head.nif, sizeof(unsigned int), 1, file);
    fread(&Head.nsub, sizeof(unsigned int), 1, file);


    fread(&Head.machine_id, sizeof(int), 1, file);
    fread(&Head.telescope_id, sizeof(int), 1, file);
    fread(&Head.nbeam, sizeof(int), 1, file);
    fread(&Head.ibeam, sizeof(int), 1, file);
    fread(&Head.sumif, sizeof(int), 1, file);


    fread(&Head.tstart, sizeof(double), 1, file);
    fread(&Head.tsamp, sizeof(double), 1, file);
    fread(&Head.fch1, sizeof(double), 1, file);
    fread(&Head.foff, sizeof(double), 1, file);
    fread(&Head.fcen, sizeof(double), 1, file);
    fread(&Head.bwchan, sizeof(double), 1, file);


    fread(&Head.src_raj, sizeof(double), 1, file);
    fread(&Head.src_dej, sizeof(double), 1, file);
    fread(&Head.az_start, sizeof(double), 1, file);
    fread(&Head.za_start, sizeof(double), 1, file);




    fread(Head.source_name, sizeof(char), 80, file);
    fread(Head.ifstream, sizeof(char), 8, file);
    fread(Head.inpfile, sizeof(char), 8, file);


    for (int i = 0; i < 4; ++i)
    {
        int len = 0;
        fread(&len, sizeof(int), 1, file);
        Head.rawfname[i] = new char[len + 1];
        memset(Head.rawfname[i], 0, (1 + len) * sizeof(char));

        fread(Head.rawfname[i], sizeof(char), len, file);

    }

    // Close the file
    fclose(file);

    return 0;
}
//--------------------------------------------------
int writeOptions(const char* strFilePath, const int noverlap, const int nbin, const int ndec, const char* obsid
    , const int part, const int device)
{
    FILE* file = fopen(strFilePath, "wb");

    if (file == NULL) {
        perror("Error opening file");
        return 1;
    }
    // Write the variables to the file
    fwrite(&noverlap, sizeof(int), 1, file);
    fwrite(&nbin, sizeof(int), 1, file);//
    fwrite(&ndec, sizeof(int), 1, file);//
    int len = strlen(obsid);
    fwrite(&len, sizeof(int), 1, file);
    fwrite(obsid, sizeof(char), len, file);
    fwrite(&part, sizeof(int), 1, file);
    fwrite(&device, sizeof(int), 1, file);
    fclose(file);
    return 0;

}
//-------------------------------------

int readOptions(const char* strFilePath, int& noverlap, int& nbin, int& ndec, char* obsid
    , int& part, int& device)
{
    // Open file for reading in binary mode
    FILE* file = fopen(strFilePath, "rb");

    if (file == NULL) {
        perror("Error opening file");
        return 1;
    }
    // Write the variables to the file
    fread(&noverlap, sizeof(int), 1, file);
    fread(&nbin, sizeof(int), 1, file);//
    //fread(&ndec, sizeof(int), 1, file);//
    ndec = 4;
    int len = 0;
    fread(&len, sizeof(int), 1, file);
    fread(obsid, sizeof(char), len, file);
    fread(&part, sizeof(int), 1, file);
    fread(&device, sizeof(int), 1, file);


    // Close the file
    fclose(file);

    return 0;
}